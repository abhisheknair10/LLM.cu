#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void kernel_standard_tiled_gemm(
    float *O, float *X, float *Transform, int m, int n, int k, int TILE_SIZE) {
    /*
        - m represents the independent dimension of the input matrix
        - n represents the independent dimenion of the transformation matrix
        - k represents the common dimension of the 2 matrices
        - Within each kernel, the output is computed as: O = matmul(X, Transform)
        - Transposing the transformation tensor is not required as virtual indexing allows
          for intended navigation along rows and columns of either tensors
        - Order of variables within kernels obey order of computation
    */
    // Kernel start
    //
    extern __shared__ float shared_mem[];
    float *X_shmem = shared_mem;
    float *T_shmem = shared_mem + TILE_SIZE * TILE_SIZE;

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    // Loop over tiles
    float value = 0.0f;
    for (int t = 0; t < ((k + TILE_SIZE - 1) / TILE_SIZE); ++t) {
        // Load tile of X into shared memory
        if (row < m && (t * TILE_SIZE + threadIdx.x) < k) {
            int X_idx = row * k + t * TILE_SIZE + threadIdx.x;
            X_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = X[X_idx];
        } else {
            X_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = 0.0f;
        }

        // Load tile of Transform into shared memory
        if (col < n && (t * TILE_SIZE + threadIdx.x) < k) {
            int T_idx = col * k + t * TILE_SIZE + threadIdx.x;
            T_shmem[threadIdx.x * TILE_SIZE + threadIdx.y] = Transform[T_idx];
        } else {
            T_shmem[threadIdx.x * TILE_SIZE + threadIdx.y] = 0.0f;
        }
        __syncthreads();

        // Compute partial sums
        for (int i = 0; i < TILE_SIZE; ++i) {
            value += X_shmem[threadIdx.y * TILE_SIZE + i] * T_shmem[i * TILE_SIZE + threadIdx.x];
        }
        __syncthreads();
    }

    // Write the result to global memory
    if (row < m && col < n) {
        O[row * n + col] = value;
    }

    return;
}

int main() {
    int m = 2;
    int n = 3;
    int k = 3;

    float X[m * k] = {1.0f, 2.0f, 3.0f,
                      4.0f, 5.0f, 6.0f};

    float Transform[n * k] = {1.0f, 2.0f, 3.0f,
                              4.0f, 5.0f, 6.0f,
                              7.0f, 8.0f, 9.0f};

    float Output[m * n] = {0.0f, 0.0f, 0.0f,
                           0.0f, 0.0f, 0.0f};

    float *d_X, *d_Transform, *d_Output;

    hipMalloc((void **)&d_X, m * k * sizeof(float));
    hipMalloc((void **)&d_Transform, n * k * sizeof(float));
    hipMalloc((void **)&d_Output, m * n * sizeof(float));

    hipMemcpy(d_X, X, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Transform, Transform, n * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Output, Output, m * n * sizeof(float), hipMemcpyHostToDevice);

    int TILE_SIZE = 32;
    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid(
        (n + TILE_SIZE - 1) / TILE_SIZE,
        (m + TILE_SIZE - 1) / TILE_SIZE);

    size_t shared_mem = 2 * TILE_SIZE * TILE_SIZE * sizeof(float);
    kernel_standard_tiled_gemm<<<grid, block, shared_mem>>>(
        d_Output, d_X, d_Transform,
        m, n, k, TILE_SIZE);
    hipDeviceSynchronize();

    hipMemcpy(Output, d_Output, m * n * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%f, ", Output[i * j]);
        }
        printf("\n");
    }

    return 0;
}