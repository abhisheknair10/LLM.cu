#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "inference.cuh"
#include "llama3/llama3.cuh"

#define CHECK_CUDA_ERROR()                                       \
    {                                                            \
        hipError_t err = hipGetLastError();                    \
        if (err != hipSuccess) {                                \
            printf("CUDA error: %s in file '%s' in line %i\n",   \
                   hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                  \
        }                                                        \
    }

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define MAX(a, b) ((a) > (b) ? (a) : (b))

const int THREADS_PER_BLOCK = 1024;

__constant__ int EMBED_SIZE;

__device__ int d_NUM_TOKENS;
int h_NUM_TOKENS;

// Allocate global mem cache on device
float *create_gmemcache(size_t mem_len, size_t type_size) {
    float *d_gcache;

    hipMalloc(&d_gcache, mem_len * type_size);

    return d_gcache;
}

void free_tensor_cuda(Tensor *t) {
    hipFree(t->d_ndim);
    hipFree(t->d_mem_len);
    hipFree(t->d_shape);
    hipFree(t->d_fp16_tensor);

    return;
}

// Print CUDA memory info
void printCudaMemoryInfo() {
    size_t free_memory = 0;
    size_t total_memory = 0;

    // Get the amount of free and total memory on the GPU
    hipError_t err = hipMemGetInfo(&free_memory, &total_memory);

    if (err == hipSuccess) {
        // Convert memory sizes from bytes to megabytes (MB)
        printf("Free GPU Memory: %.2f MB\n", (float)free_memory / (1024 * 1024));
        printf("Total GPU Memory: %.2f MB\n", (float)total_memory / (1024 * 1024));
    } else {
        printf("Failed to get CUDA memory info: %s\n", hipGetErrorString(err));
    }

    return;
}

// Kernel to check and print the embeddings
__global__ void check_embedding(__half *fp16_tensor) {
    for (int token_idx = 0; token_idx < d_NUM_TOKENS; token_idx++) {
        printf("Token %d embeddings:\n", token_idx + 1);
        for (int i = 0; i < EMBED_SIZE; i++) {
            float embedding = __half2float(fp16_tensor[token_idx * EMBED_SIZE + i]);
            printf("%f ", embedding);
        }
        printf("\n\n\n\n\n");
    }

    return;
}

/* ******************************** Inference Code ******************************** */
void inference(Llama3 *llama3_model, Tensor *X, int *d_tokens, int *h_tokens) {
    int embed_size = 4096;
    hipMemcpyToSymbol(HIP_SYMBOL(EMBED_SIZE), &embed_size, sizeof(int));

    // Set NUM_TOKENS value in device memory
    h_NUM_TOKENS = h_tokens[0] - 1;
    hipMemcpyToSymbol(HIP_SYMBOL(d_NUM_TOKENS), &h_NUM_TOKENS, sizeof(int));
    free(h_tokens);

    tokens_to_embeddings(X, llama3_model, d_tokens);

    // Ahead Of Time memory allocations
    // Allocate once, use everywhere
    Tensor *PN_X = (Tensor *)malloc(sizeof(Tensor));
    _create_intermediary_prenorm_tensor_copy(PN_X, X);

    float *d_gcache = create_gmemcache(10000, sizeof(float));

    Tensor *Q = (Tensor *)malloc(sizeof(Tensor));
    Tensor *K = (Tensor *)malloc(sizeof(Tensor));
    Tensor *V = (Tensor *)malloc(sizeof(Tensor));
    _create_intermediary_attention_tensor(Q, llama3_model->layers[0]->self_attn_q_proj);
    _create_intermediary_attention_tensor(K, llama3_model->layers[0]->self_attn_k_proj);
    _create_intermediary_attention_tensor(V, llama3_model->layers[0]->self_attn_v_proj);

    // Run Inference
    for (int i = 0; i < llama3_model->n_layers; i++) {
        // Pre-attention normalization
        copy_fp16_tensor(PN_X, X);
        compute_layer_norm(llama3_model->layers[i]->input_layernorm, X, d_gcache);

        // Attention computation
        compute_qkv_tensors(Q, K, V, llama3_model->layers[i], X);

        break;
    }

    printCudaMemoryInfo();

    free_tensor_cuda(PN_X);
    free_tensor_cuda(Q);
    free_tensor_cuda(K);
    free_tensor_cuda(V);
    hipFree(d_gcache);

    return;
}

/* *************************** Convert Tokens to Embeddings *************************** */
void tokens_to_embeddings(Tensor *X, Llama3 *llama3_model, int *d_tokens) {
    // Order threads into blocks
    int total_threads = *(X->mem_len);
    int blocks = (total_threads + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    kernel_tokens_to_embeddings<<<blocks, THREADS_PER_BLOCK>>>(
        X->d_fp16_tensor, llama3_model->embed_tokens->d_fp16_tensor, d_tokens);

    hipDeviceSynchronize();

    // check_embedding<<<1, 1>>>(X->d_fp16_tensor);
    // hipDeviceSynchronize();

    return;
}

__global__ void kernel_tokens_to_embeddings(__half *X_tensor, __half *Embed, int *tokens) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int total_elements = d_NUM_TOKENS * EMBED_SIZE;

    if (idx >= total_elements) return;

    int token_idx = idx / EMBED_SIZE;
    int embed_idx = idx % EMBED_SIZE;

    X_tensor[(token_idx * EMBED_SIZE) + embed_idx] =
        Embed[(tokens[token_idx + 1] * EMBED_SIZE) + embed_idx];

    return;
}

/* ******************************* Layer Normalization ******************************* */
void _create_intermediary_prenorm_tensor_copy(Tensor *Y, Tensor *X) {
    int *d_ndim;
    int *d_mem_len;
    int *d_shape;
    __half *d_fp16_tensor;

    Y->ndim = (int *)malloc(sizeof(int));
    *(Y->ndim) = *(X->ndim);

    Y->mem_len = (int *)malloc(sizeof(int));
    *(Y->mem_len) = *(X->mem_len);

    Y->shape = (int *)malloc(sizeof(int) * (*(X->ndim)));
    for (int i = 0; i < (*(X->ndim)); i++) {
        Y->shape[i] = X->shape[i];
    }

    // Allocate CUDA memory
    hipMalloc(&d_ndim, sizeof(int));
    hipMalloc(&d_mem_len, sizeof(int));
    hipMalloc(&d_shape, sizeof(int) * (*(Y->ndim)));
    hipMalloc(&d_fp16_tensor, sizeof(__half) * (*(Y->mem_len)));

    // Copy data to device
    hipMemcpy(d_ndim, Y->ndim, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mem_len, Y->mem_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_shape, Y->shape, sizeof(int) * (*(Y->ndim)), hipMemcpyHostToDevice);

    // Assign device pointers
    Y->d_ndim = d_ndim;
    Y->d_mem_len = d_mem_len;
    Y->d_shape = d_shape;
    Y->d_fp16_tensor = d_fp16_tensor;

    return;
}

void copy_fp16_tensor(Tensor *Y, Tensor *X) {
    hipMemcpy(
        Y->d_fp16_tensor,
        X->d_fp16_tensor,
        sizeof(__half) * (*(Y->mem_len)),
        hipMemcpyDeviceToDevice);

    return;
}

void compute_layer_norm(Tensor *RMSNorm, Tensor *X, float *d_gcache) {
    int blocks_x = 4096 / THREADS_PER_BLOCK;
    int blocks_y = h_NUM_TOKENS;

    dim3 blocks(blocks_x, blocks_y);
    size_t shared_mem_size = THREADS_PER_BLOCK * sizeof(float);

    kernel_compute_rms_norm<<<blocks, THREADS_PER_BLOCK, shared_mem_size>>>(
        X->d_fp16_tensor, RMSNorm->d_fp16_tensor, d_gcache);
    hipDeviceSynchronize();

    kernel_compute_norm_tensor<<<blocks, THREADS_PER_BLOCK>>>(
        X->d_fp16_tensor, RMSNorm->d_fp16_tensor, d_gcache);
    hipDeviceSynchronize();

    // check_embedding<<<1, 1>>>(X->d_fp16_tensor);
    // hipDeviceSynchronize();
}

__global__ void kernel_compute_rms_norm(__half *X_tensor, __half *RMSNorm_tensor, float *d_gcache) {
    extern __shared__ float shared_mem[];

    int token_idx = blockIdx.y;
    int embed_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (token_idx >= d_NUM_TOKENS) return;
    if (embed_idx >= EMBED_SIZE) return;

    // Convert __half to float and square
    float x = __half2float(X_tensor[(token_idx * EMBED_SIZE) + embed_idx]);
    shared_mem[threadIdx.x] = x * x;
    __syncthreads();

    // Perform parallel reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shared_mem[threadIdx.x] += shared_mem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Store partial sums in d_gcache
    if (threadIdx.x == 0) {
        d_gcache[blockIdx.y * gridDim.x + blockIdx.x] = shared_mem[0];
    }
    __syncthreads();

    float rms = 0.0f;
    float eps = 1e-6f;

    // Compute the RMS value
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int i = 0; i < gridDim.x; i++) {
            rms += d_gcache[blockIdx.y * gridDim.x + i];
        }
        rms = sqrtf((rms + eps) / (float)EMBED_SIZE);
        d_gcache[blockIdx.y] = rms;
    }

    return;
}

__global__ void kernel_compute_norm_tensor(__half *X_tensor, __half *RMSNorm_tensor, float *d_gcache) {
    int token_idx = blockIdx.y;
    int embed_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (token_idx >= d_NUM_TOKENS) return;
    if (embed_idx >= EMBED_SIZE) return;

    // Normalize the input and write back
    float rms = d_gcache[blockIdx.y];
    float x = __half2float(X_tensor[(token_idx * EMBED_SIZE) + embed_idx]);
    float scale = __half2float(RMSNorm_tensor[embed_idx]);

    float res = (x / rms) * scale;
    X_tensor[(token_idx * EMBED_SIZE) + embed_idx] = __float2half(res);

    return;
}

/* ******************************* Attention Computation ******************************* */
void _create_intermediary_attention_tensor(Tensor *Attention_Tensor, Tensor *Linear) {
    int *d_ndim;
    int *d_mem_len;
    int *d_shape;
    __half *d_fp16_tensor;

    Attention_Tensor->ndim = (int *)malloc(sizeof(int));
    *(Attention_Tensor->ndim) = 2;

    Attention_Tensor->mem_len = (int *)malloc(sizeof(int));
    *(Attention_Tensor->mem_len) = Linear->shape[0] * h_NUM_TOKENS;

    Attention_Tensor->shape = (int *)malloc(sizeof(int) * 2);
    Attention_Tensor->shape[0] = h_NUM_TOKENS;
    Attention_Tensor->shape[1] = Linear->shape[0];

    // Allocate CUDA memory
    hipMalloc(&d_ndim, sizeof(int));
    hipMalloc(&d_mem_len, sizeof(int));
    hipMalloc(&d_shape, sizeof(int) * 2);
    hipMalloc(&d_fp16_tensor, sizeof(__half) * (*(Attention_Tensor->mem_len)));

    // Copy data to device
    hipMemcpy(d_ndim, Attention_Tensor->ndim, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mem_len, Attention_Tensor->mem_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_shape, Attention_Tensor->shape, sizeof(int) * 2, hipMemcpyHostToDevice);

    // Assign device pointers
    Attention_Tensor->d_ndim = d_ndim;
    Attention_Tensor->d_mem_len = d_mem_len;
    Attention_Tensor->d_shape = d_shape;
    Attention_Tensor->d_fp16_tensor = d_fp16_tensor;

    return;
}

void compute_qkv_tensors(Tensor *Q, Tensor *K, Tensor *V, Llama3Layer *L3_Layer, Tensor *X) {
    /*
    // Compute Queries
    kernel_compute_attention_tensors<<<4, 1024>>>(
        Q->d_fp16_tensor, Q->d_ndim, Q->d_shape,
        L3_Layer->self_attn_q_proj->d_fp16_tensor, L3_Layer->self_attn_q_proj->d_ndim, L3_Layer->self_attn_q_proj->d_shape,
        X->d_fp16_tensor, X->d_ndim, X->d_shape);
    hipDeviceSynchronize();
    */

    return;
}

__global__ void kernel_compute_attention_tensors(
    __half *O_tensor, int *O_ndim, int *O_shape,
    __half *Linear_tensor, int *Linear_ndim, int *Linear_shape,
    __half *X_tensor, int *X_ndim, int *X_shape) {
    // Start of kernel

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int dim = gridDim.x * blockDim.x;

    // [4096][1024, 4096]

    if (idx >= dim) return;

    __half sum = 0;
    for (int i = 0; i < EMBED_SIZE; i++) {
        sum += (X_tensor[i] * Linear_tensor[EMBED_SIZE * i]);
    }

    O_tensor[idx] = sum;

    return;
}