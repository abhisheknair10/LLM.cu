#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "inference.cuh"
#include "llama3/llama3.cuh"

#define CHECK_CUDA_ERROR()                                       \
    {                                                            \
        hipError_t err = hipGetLastError();                    \
        if (err != hipSuccess) {                                \
            printf("CUDA error: %s in file '%s' in line %i\n",   \
                   hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                  \
        }                                                        \
    }

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define MAX(a, b) ((a) > (b) ? (a) : (b))

const int THREADS_PER_BLOCK = 1024;

__constant__ int EMBED_SIZE = 4096;

__device__ int d_NUM_TOKENS;
int h_NUM_TOKENS;

void printCudaMemoryInfo() {
    size_t free_memory = 0;
    size_t total_memory = 0;

    // Get the amount of free and total memory on the GPU
    hipError_t err = hipMemGetInfo(&free_memory, &total_memory);

    if (err == hipSuccess) {
        // Convert memory sizes from bytes to megabytes (MB)
        printf("Free GPU Memory: %.2f MB\n", (float)free_memory / (1024 * 1024));
        printf("Total GPU Memory: %.2f MB\n", (float)total_memory / (1024 * 1024));
    } else {
        printf("Failed to get CUDA memory info: %s\n", hipGetErrorString(err));
    }
}

// Kernel to check and print the embeddings
__global__ void check_embedding(__half *fp16_tensor) {
    for (int token_idx = 0; token_idx < d_NUM_TOKENS; token_idx++) {
        printf("Token %d embeddings:\n", token_idx + 1);
        for (int i = 0; i < EMBED_SIZE; i++) {
            float embedding = __half2float(fp16_tensor[token_idx * EMBED_SIZE + i]);
            printf("%f ", embedding);
        }
        printf("\n\n\n\n\n");
    }

    return;
}

/* ******************************** Inference Code ******************************** */

void inference(Llama3 *llama3_model, Tensor *X, int *d_tokens, int *h_tokens) {
    // Set NUM_TOKENS value in device memory
    h_NUM_TOKENS = h_tokens[0] - 1;
    hipMemcpyToSymbol(HIP_SYMBOL(d_NUM_TOKENS), &h_NUM_TOKENS, sizeof(int));
    free(h_tokens);

    tokens_to_embeddings(X, llama3_model, d_tokens);

    // Ahead Of Time memory allocations
    // Allocate once, use everywhere
    Tensor *Q, *K, *V;
    _create_intermediary_attention_tensor(&Q, llama3_model->layers[0]->self_attn_q_proj);
    _create_intermediary_attention_tensor(&K, llama3_model->layers[0]->self_attn_k_proj);
    _create_intermediary_attention_tensor(&V, llama3_model->layers[0]->self_attn_v_proj);

    // Run Inference
    for (int i = 0; i < llama3_model->n_layers; i++) {
        compute_qkv_tensors(Q, K, V, llama3_model->layers[i], X);
    }

    printCudaMemoryInfo();
}

void tokens_to_embeddings(Tensor *X, Llama3 *llama3_model, int *d_tokens) {
    // Order threads into blocks
    int blocks = (h_NUM_TOKENS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    kernel_tokens_to_embeddings<<<blocks, THREADS_PER_BLOCK>>>(
        X->d_fp16_tensor, llama3_model->embed_tokens->d_fp16_tensor, d_tokens);

    hipDeviceSynchronize();

    // check_embedding<<<1, 1>>>(X->d_fp16_tensor);
    // hipDeviceSynchronize();
}

__global__ void kernel_tokens_to_embeddings(__half *fp16_tensor, __half *Embed, int *tokens) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // tokens[0] consists of the length of the entire tokens array
    if (idx > 0 && idx <= d_NUM_TOKENS) {
        int managed_offset = idx - 1;
        for (int i = 0; i < EMBED_SIZE; i++) {
            fp16_tensor[(managed_offset * EMBED_SIZE) + i] =
                Embed[(tokens[idx] * EMBED_SIZE) + i];
        }
    }

    return;
}

void _create_intermediary_attention_tensor(Tensor *Attention_Tensor, Tensor *Linear) {
    // Allocate the Tensor struct
    Attention_Tensor = (Tensor *)malloc(sizeof(Tensor));

    int *d_ndim;
    int *d_mem_len;
    int *d_shape;
    __half *d_fp16_tensor;

    Attention_Tensor->ndim = (int *)malloc(sizeof(int));
    *(Attention_Tensor->ndim) = *(Linear->ndim);

    Attention_Tensor->mem_len = (int *)malloc(sizeof(int));
    *(Attention_Tensor->mem_len) = *(Linear->mem_len);

    (*Attention_Tensor)->shape = (int *)malloc(sizeof(int) * (*(Attention_Tensor->ndim)));
    for (int i = 0; i < *(*Attention_Tensor)->ndim; i++) {
        Attention_Tensor->shape[i] = Linear->shape[i];
    }

    // Allocate CUDA memory
    hipMalloc((void **)&d_ndim, sizeof(int));
    hipMalloc((void **)&d_mem_len, sizeof(int));
    hipMalloc((void **)&d_shape, sizeof(int) * (*(Attention_Tensor->ndim)));
    hipMalloc((void **)&d_fp16_tensor, sizeof(__half) * (*(Attention_Tensor->mem_len)));

    // Copy data to device
    hipMemcpy(d_ndim, Attention_Tensor->ndim, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mem_len, Attention_Tensor->mem_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_shape, Attention_Tensor->shape, sizeof(int) * (*(Attention_Tensor->ndim)), hipMemcpyHostToDevice);

    // Assign device pointers
    Attention_Tensor->d_ndim = d_ndim;
    Attention_Tensor->d_mem_len = d_mem_len;
    Attention_Tensor->d_shape = d_shape;
    Attention_Tensor->d_fp16_tensor = d_fp16_tensor;

    return;
}

void compute_qkv_tensors(Tensor *Q, Tensor *K, Tensor *V, Llama3Layer *L3_Layer, Tensor *X) {
    // Compute Queries
    kernel_compute_attention_tensors<<<1, 1>>>(
        Q, L3_Layer->self_attn_q_proj, X->d_fp16_tensor);

    hipDeviceSynchronize();

    // Compute Keys
    kernel_compute_attention_tensors<<<1, 1>>>(
        K, L3_Layer->self_attn_k_proj, X->d_fp16_tensor);

    hipDeviceSynchronize();

    // Compute Values
    kernel_compute_attention_tensors<<<1, 1>>>(
        V, L3_Layer->self_attn_v_proj, X->d_fp16_tensor);

    hipDeviceSynchronize();
}

__global__ void kernel_compute_attention_tensors(Tensor *O, __half *Linear, __half *X) {
    // idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < (*(O->ndim)); i++) {
        printf("%d, \n", O->shape[i]);
    }
}