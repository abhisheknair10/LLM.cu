#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "inference.cuh"
#include "llama3/llama3.cuh"

#define CHECK_CUDA_ERROR()                                       \
    {                                                            \
        hipError_t err = hipGetLastError();                    \
        if (err != hipSuccess) {                                \
            printf("CUDA error: %s in file '%s' in line %i\n",   \
                   hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                  \
        }                                                        \
    }

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define MAX(a, b) ((a) > (b) ? (a) : (b))

const int MAX_THREADS_PER_BLOCK = 1024;

__constant__ int EMBED_SIZE;

__device__ int d_NUM_TOKENS;
int h_NUM_TOKENS;

// Allocate global mem cache on device
float *create_gmemcache(size_t mem_len, size_t type_size) {
    float *d_gcache;
    hipMalloc(&d_gcache, mem_len * type_size);
    return d_gcache;
}

void free_tensor_cuda(Tensor *t) {
    hipFree(t->d_ndim);
    hipFree(t->d_mem_len);
    hipFree(t->d_shape);
    hipFree(t->d_fp16_tensor);
    return;
}

// Print CUDA memory info
void printCudaMemoryInfo() {
    size_t free_memory = 0;
    size_t total_memory = 0;
    hipError_t err = hipMemGetInfo(&free_memory, &total_memory);
    if (err == hipSuccess) {
        printf("Free GPU Memory: %.2f MB\n", (float)free_memory / (1024 * 1024));
        printf("Total GPU Memory: %.2f MB\n", (float)total_memory / (1024 * 1024));
    } else {
        printf("Failed to get CUDA memory info: %s\n", hipGetErrorString(err));
    }
    return;
}

// Kernel to check and print the embeddings
__global__ void check_embedding(__half *fp16_tensor, int dim) {
    for (int token_idx = 0; token_idx < d_NUM_TOKENS; token_idx++) {
        printf("Token %d embeddings:\n", token_idx + 1);
        for (int i = 0; i < dim; i++) {
            float embedding = __half2float(fp16_tensor[token_idx * dim + i]);
            printf("%f ", embedding);
        }
        printf("\n\n\n\n\n");
    }
    return;
}

/* ******************************** Inference Code ******************************** */
void inference(Llama3 *llama3_model, Tensor *X, int *d_tokens, int *h_tokens) {
    int embed_size = 4096;
    hipMemcpyToSymbol(HIP_SYMBOL(EMBED_SIZE), &embed_size, sizeof(int));

    h_NUM_TOKENS = h_tokens[0] - 1;
    hipMemcpyToSymbol(HIP_SYMBOL(d_NUM_TOKENS), &h_NUM_TOKENS, sizeof(int));
    free(h_tokens);

    tokens_to_embeddings(X, llama3_model, d_tokens);

    Tensor *PN_X = (Tensor *)malloc(sizeof(Tensor));
    _create_intermediary_prenorm_tensor_copy(PN_X, X);

    int total_blocks_x = (embed_size + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;
    size_t cache_size_K = h_NUM_TOKENS * llama3_model->layers[0]->self_attn_k_proj->shape[0] * total_blocks_x;
    size_t cache_size_V = h_NUM_TOKENS * llama3_model->layers[0]->self_attn_v_proj->shape[0] * total_blocks_x;
    size_t cache_size_Q = h_NUM_TOKENS * llama3_model->layers[0]->self_attn_q_proj->shape[0] * total_blocks_x;

    size_t cache_offset_K = 0;
    size_t cache_offset_V = cache_size_K;
    size_t cache_offset_Q = cache_size_K + cache_size_V;

    size_t required_cache_size = cache_size_K + cache_size_V + cache_size_Q;
    float *d_gcache = create_gmemcache(required_cache_size, sizeof(float));

    Tensor *Q = (Tensor *)malloc(sizeof(Tensor));
    Tensor *K = (Tensor *)malloc(sizeof(Tensor));
    Tensor *V = (Tensor *)malloc(sizeof(Tensor));
    _create_intermediary_attention_tensor(Q, llama3_model->layers[0]->self_attn_q_proj);
    _create_intermediary_attention_tensor(K, llama3_model->layers[0]->self_attn_k_proj);
    _create_intermediary_attention_tensor(V, llama3_model->layers[0]->self_attn_v_proj);

    for (int i = 0; i < llama3_model->n_layers; i++) {
        copy_fp16_tensor(PN_X, X);
        compute_layer_norm(llama3_model->layers[i]->input_layernorm, X, d_gcache);
        compute_qkv_tensors(Q, K, V, llama3_model->layers[i], X, d_gcache,
                            cache_offset_Q, cache_offset_K, cache_offset_V);
        break;
    }

    printCudaMemoryInfo();

    free_tensor_cuda(PN_X);
    free_tensor_cuda(Q);
    free_tensor_cuda(K);
    free_tensor_cuda(V);
    hipFree(d_gcache);

    return;
}

/* *************************** Convert Tokens to Embeddings *************************** */
void tokens_to_embeddings(Tensor *X, Llama3 *llama3_model, int *d_tokens) {
    int total_threads = *(X->mem_len);
    int blocks = (total_threads + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;

    kernel_tokens_to_embeddings<<<blocks, MAX_THREADS_PER_BLOCK>>>(
        X->d_fp16_tensor, llama3_model->embed_tokens->d_fp16_tensor, d_tokens);

    hipDeviceSynchronize();
    return;
}

__global__ void kernel_tokens_to_embeddings(__half *X_tensor, __half *Embed, int *tokens) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = d_NUM_TOKENS * EMBED_SIZE;
    if (idx >= total_elements) return;
    int token_idx = idx / EMBED_SIZE;
    int embed_idx = idx % EMBED_SIZE;
    X_tensor[(token_idx * EMBED_SIZE) + embed_idx] =
        Embed[(tokens[token_idx + 1] * EMBED_SIZE) + embed_idx];
    return;
}

/* ******************************* Layer Normalization ******************************* */
void _create_intermediary_prenorm_tensor_copy(Tensor *Y, Tensor *X) {
    int *d_ndim;
    int *d_mem_len;
    int *d_shape;
    __half *d_fp16_tensor;

    Y->ndim = (int *)malloc(sizeof(int));
    *(Y->ndim) = *(X->ndim);

    Y->mem_len = (int *)malloc(sizeof(int));
    *(Y->mem_len) = *(X->mem_len);

    Y->shape = (int *)malloc(sizeof(int) * (*(X->ndim)));
    for (int i = 0; i < (*(X->ndim)); i++) {
        Y->shape[i] = X->shape[i];
    }

    hipMalloc(&d_ndim, sizeof(int));
    hipMalloc(&d_mem_len, sizeof(int));
    hipMalloc(&d_shape, sizeof(int) * (*(Y->ndim)));
    hipMalloc(&d_fp16_tensor, sizeof(__half) * (*(Y->mem_len)));

    hipMemcpy(d_ndim, Y->ndim, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mem_len, Y->mem_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_shape, Y->shape, sizeof(int) * (*(Y->ndim)), hipMemcpyHostToDevice);

    Y->d_ndim = d_ndim;
    Y->d_mem_len = d_mem_len;
    Y->d_shape = d_shape;
    Y->d_fp16_tensor = d_fp16_tensor;

    return;
}

void copy_fp16_tensor(Tensor *Y, Tensor *X) {
    hipMemcpy(
        Y->d_fp16_tensor,
        X->d_fp16_tensor,
        sizeof(__half) * (*(Y->mem_len)),
        hipMemcpyDeviceToDevice);
    return;
}

void compute_layer_norm(Tensor *RMSNorm, Tensor *X, float *d_gcache) {
    int blocks_x = (EMBED_SIZE + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;
    int blocks_y = h_NUM_TOKENS;

    dim3 blocks(blocks_x, blocks_y);
    size_t shared_mem_size = MAX_THREADS_PER_BLOCK * sizeof(float);

    kernel_compute_rms_norm<<<blocks, MAX_THREADS_PER_BLOCK, shared_mem_size>>>(
        X->d_fp16_tensor, RMSNorm->d_fp16_tensor, d_gcache);
    hipDeviceSynchronize();

    kernel_compute_norm_tensor<<<blocks, MAX_THREADS_PER_BLOCK>>>(
        X->d_fp16_tensor, RMSNorm->d_fp16_tensor, d_gcache);
    hipDeviceSynchronize();
}

__global__ void kernel_compute_rms_norm(__half *X_tensor, __half *RMSNorm_tensor, float *d_gcache) {
    extern __shared__ float shared_mem[];

    int token_idx = blockIdx.y;
    int embed_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (token_idx >= d_NUM_TOKENS) return;
    if (embed_idx >= EMBED_SIZE) return;

    float x = __half2float(X_tensor[(token_idx * EMBED_SIZE) + embed_idx]);
    shared_mem[threadIdx.x] = x * x;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shared_mem[threadIdx.x] += shared_mem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        d_gcache[blockIdx.y * gridDim.x + blockIdx.x] = shared_mem[0];
    }
    __syncthreads();

    float rms = 0.0f;
    float eps = 1e-6f;

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int i = 0; i < gridDim.x; i++) {
            rms += d_gcache[blockIdx.y * gridDim.x + i];
        }
        rms = sqrtf((rms + eps) / (float)EMBED_SIZE);
        d_gcache[blockIdx.y] = rms;
    }

    return;
}

__global__ void kernel_compute_norm_tensor(__half *X_tensor, __half *RMSNorm_tensor, float *d_gcache) {
    int token_idx = blockIdx.y;
    int embed_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (token_idx >= d_NUM_TOKENS) return;
    if (embed_idx >= EMBED_SIZE) return;

    float rms = d_gcache[blockIdx.y];
    float x = __half2float(X_tensor[(token_idx * EMBED_SIZE) + embed_idx]);
    float scale = __half2float(RMSNorm_tensor[embed_idx]);

    float res = (x / rms) * scale;
    X_tensor[(token_idx * EMBED_SIZE) + embed_idx] = __float2half(res);

    return;
}

/* ******************************* Attention Computation ******************************* */
void _create_intermediary_attention_tensor(Tensor *Attention_Tensor, Tensor *Linear) {
    int *d_ndim;
    int *d_mem_len;
    int *d_shape;
    __half *d_fp16_tensor;

    Attention_Tensor->ndim = (int *)malloc(sizeof(int));
    *(Attention_Tensor->ndim) = 2;

    Attention_Tensor->mem_len = (int *)malloc(sizeof(int));
    *(Attention_Tensor->mem_len) = Linear->shape[0] * h_NUM_TOKENS;

    Attention_Tensor->shape = (int *)malloc(sizeof(int) * 2);
    Attention_Tensor->shape[0] = h_NUM_TOKENS;
    Attention_Tensor->shape[1] = Linear->shape[0];

    hipMalloc(&d_ndim, sizeof(int));
    hipMalloc(&d_mem_len, sizeof(int));
    hipMalloc(&d_shape, sizeof(int) * 2);
    hipMalloc(&d_fp16_tensor, sizeof(__half) * (*(Attention_Tensor->mem_len)));

    hipMemcpy(d_ndim, Attention_Tensor->ndim, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mem_len, Attention_Tensor->mem_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_shape, Attention_Tensor->shape, sizeof(int) * 2, hipMemcpyHostToDevice);

    Attention_Tensor->d_ndim = d_ndim;
    Attention_Tensor->d_mem_len = d_mem_len;
    Attention_Tensor->d_shape = d_shape;
    Attention_Tensor->d_fp16_tensor = d_fp16_tensor;

    return;
}

void compute_qkv_tensors(Tensor *Q, Tensor *K, Tensor *V,
                         Llama3Layer *L3_Layer, Tensor *X, float *d_gcache,
                         size_t cache_offset_Q, size_t cache_offset_K, size_t cache_offset_V) {

    _abstract_intermediate_attensor_kernel_call(L3_Layer->self_attn_k_proj, X, d_gcache, cache_offset_K);
    _abstract_intermediate_attensor_kernel_call(L3_Layer->self_attn_v_proj, X, d_gcache, cache_offset_V);
    _abstract_intermediate_attensor_kernel_call(L3_Layer->self_attn_q_proj, X, d_gcache, cache_offset_Q);

    hipDeviceSynchronize();

    _abstract_full_attensor_kernel_call(K, L3_Layer->self_attn_k_proj, d_gcache, cache_offset_K);
    _abstract_full_attensor_kernel_call(V, L3_Layer->self_attn_v_proj, d_gcache, cache_offset_V);
    _abstract_full_attensor_kernel_call(Q, L3_Layer->self_attn_q_proj, d_gcache, cache_offset_Q);

    hipDeviceSynchronize();

    check_embedding<<<1, 1>>>(Q->d_fp16_tensor, Q->shape[1]);
    hipDeviceSynchronize();
    printf("Queries\n");
    check_embedding<<<1, 1>>>(K->d_fp16_tensor, K->shape[1]);
    hipDeviceSynchronize();
    printf("Keys\n");
    check_embedding<<<1, 1>>>(V->d_fp16_tensor, V->shape[1]);
    hipDeviceSynchronize();
    printf("Values\n");

    CHECK_CUDA_ERROR();

    return;
}

void _abstract_intermediate_attensor_kernel_call(Tensor *Proj_Layer, Tensor *X,
                                                 float *d_gcache, size_t cache_offset) {
    int embed_size = EMBED_SIZE;
    int out_features = Proj_Layer->shape[0];

    int blockx = (embed_size + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;
    int blocky = (out_features + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;
    int blockz = h_NUM_TOKENS;

    dim3 blocks(blockx, blocky, blockz);
    size_t shared_mem_size = MAX_THREADS_PER_BLOCK * sizeof(float);

    kernel_compute_intermediate_attention_matmul<<<blocks, MAX_THREADS_PER_BLOCK, shared_mem_size>>>(
        Proj_Layer->d_fp16_tensor, Proj_Layer->d_shape,
        X->d_fp16_tensor, d_gcache, cache_offset,
        embed_size, out_features);
}

__global__ void kernel_compute_intermediate_attention_matmul(
    __half *Linear_tensor, int *Linear_shape,
    __half *X_tensor, float *d_gcache, size_t cache_offset,
    int embed_size, int out_features) {

    extern __shared__ float shared_mem[];

    int total_blocks_x = (embed_size + blockDim.x - 1) / blockDim.x;

    int token_idx = blockIdx.z;
    int f_block_idx = blockIdx.y;
    int e_block_idx = blockIdx.x;
    int thread_idx = threadIdx.x;

    int embed_idx = e_block_idx * blockDim.x + thread_idx;
    int fcoord_idx = f_block_idx * blockDim.x + thread_idx;

    if (embed_idx >= embed_size || fcoord_idx >= out_features)
        return;

    float x = __half2float(X_tensor[token_idx * embed_size + embed_idx]);
    float f = __half2float(Linear_tensor[fcoord_idx * embed_size + embed_idx]);
    shared_mem[thread_idx] = x * f;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (thread_idx < stride) {
            shared_mem[thread_idx] += shared_mem[thread_idx + stride];
        }
        __syncthreads();
    }

    if (thread_idx == 0) {
        int total_blocks_y = (out_features + blockDim.x - 1) / blockDim.x;
        int cache_idx = cache_offset +
                        token_idx * out_features * total_blocks_x +
                        f_block_idx * total_blocks_x +
                        e_block_idx;
        d_gcache[cache_idx] = shared_mem[0];
    }
}

void _abstract_full_attensor_kernel_call(Tensor *Attention_Tensor, Tensor *Proj_Layer,
                                         float *d_gcache, size_t cache_offset) {
    int out_features = Proj_Layer->shape[0];
    int total_blocks_x = (EMBED_SIZE + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;

    int blockx = (out_features + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;
    int blocky = h_NUM_TOKENS;

    dim3 blocks(blockx, blocky);

    kernel_compute_full_attention_tensors<<<blocks, MAX_THREADS_PER_BLOCK>>>(
        Attention_Tensor->d_fp16_tensor, Proj_Layer->d_shape,
        d_gcache, cache_offset,
        out_features, total_blocks_x);
}

__global__ void kernel_compute_full_attention_tensors(
    __half *O_tensor, int *Linear_shape,
    float *d_gcache, size_t cache_offset,
    int out_features, int total_blocks_x) {

    int token_idx = blockIdx.y;
    int f_block_idx = blockIdx.x;
    int thread_idx = threadIdx.x;

    int fcoord_idx = f_block_idx * blockDim.x + thread_idx;

    if (token_idx >= d_NUM_TOKENS) return;
    if (fcoord_idx >= out_features) return;

    float sum = 0.0f;
    for (int i = 0; i < total_blocks_x; i++) {
        int cache_idx = cache_offset +
                        token_idx * out_features * total_blocks_x +
                        fcoord_idx * total_blocks_x +
                        i;
        sum += d_gcache[cache_idx];
    }

    O_tensor[token_idx * out_features + fcoord_idx] = __float2half(sum);
}
