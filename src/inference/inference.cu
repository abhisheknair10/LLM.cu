#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "inference.cuh"
#include "llama3/llama3.cuh"

#define CHECK_CUDA_ERROR()                                       \
    {                                                            \
        hipError_t err = hipGetLastError();                    \
        if (err != hipSuccess) {                                \
            printf("CUDA error: %s in file '%s' in line %i\n",   \
                   hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                  \
        }                                                        \
    }

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define MAX(a, b) ((a) > (b) ? (a) : (b))

const int MAX_THREADS_PER_BLOCK = 1024;

__device__ int EMBED_SIZE;

__device__ int d_NUM_TOKENS;
int h_NUM_TOKENS;

/* ************************************ HELPERS ************************************ */
void free_tensor_cuda(Tensor *t) {
    hipFree(t->d_ndim);
    hipFree(t->d_mem_len);
    hipFree(t->d_shape);
    hipFree(t->d_fp16_tensor);

    return;
}

// Print CUDA memory info
void printCudaMemoryInfo() {
    size_t free_memory = 0;
    size_t total_memory = 0;

    // Get the amount of free and total memory on the GPU
    hipError_t err = hipMemGetInfo(&free_memory, &total_memory);

    if (err == hipSuccess) {
        // Convert memory sizes from bytes to megabytes (MB)
        printf("Free GPU Memory: %.2f MB\n", (float)free_memory / (1024 * 1024));
        printf("Total GPU Memory: %.2f MB\n", (float)total_memory / (1024 * 1024));
    } else {
        printf("Failed to get CUDA memory info: %s\n", hipGetErrorString(err));
    }

    return;
}

// Kernel to check and print the embeddings
__global__ void check_embedding(__half *fp16_tensor, int dim) {
    for (int token_idx = 0; token_idx < d_NUM_TOKENS; token_idx++) {
        printf("Token %d embeddings:\n", token_idx);
        for (int i = 0; i < dim; i++) {
            printf("%f, ", __half2float(fp16_tensor[token_idx * dim + i]));
        }
        printf("\n");
        printf("\n\n");
    }

    return;
}
/*
__global__ void check_embedding(__half *fp16_tensor, int dim) {
    for (int token_idx = 0; token_idx < d_NUM_TOKENS; token_idx++) {
        printf("Token %d embeddings:\n", token_idx + 1);
        int max = 0;
        float curr_max = 0.0f;
        for (int i = 0; i < dim; i++) {
            float embedding = __half2float(fp16_tensor[token_idx * dim + i]);

            if (embedding > curr_max) {
                curr_max = embedding;
                max = i;
            }
        }
        printf("%d\n", max);
        printf("\n\n");
    }

    return;
}
*/
/* ************************************* Cache ************************************* */
// Allocate global mem cache on device
void *create_gmemcache(size_t mem_len, size_t type_size) {
    void *d_gcache;

    hipMalloc(&d_gcache, mem_len * type_size);

    return d_gcache;
}

CudaCache *init_cache(Llama3 *llama3_model) {
    // Ahead Of Time memory allocations
    // Allocate once, use everywhere
    CudaCache *Cache = (CudaCache *)malloc(sizeof(CudaCache));

    // Allocate Memory --------------------------------------------------------
    Tensor *PN_X = _create_intermediary_prenorm_tensor_copy();

    Tensor *Q = _create_intermediary_attention_tensor(llama3_model->layers[0]->self_attn_q_proj);
    Tensor *K = _create_intermediary_attention_tensor(llama3_model->layers[0]->self_attn_k_proj);
    Tensor *V = _create_intermediary_attention_tensor(llama3_model->layers[0]->self_attn_v_proj);

    float *d_attention_score_cache = (float *)create_gmemcache(2048 * 2048, sizeof(float));

    __half *d_feedforward_cache_gate = (__half *)create_gmemcache(14336 * 2048, sizeof(__half));
    __half *d_feedforward_cache_up = (__half *)create_gmemcache(14336 * 2048, sizeof(__half));

    __half *next_token = (__half *)create_gmemcache(128256 * 2048, sizeof(__half));

    // Save pointers to Struct --------------------------------------------------------
    Cache->PN_X = PN_X;

    Cache->Q = Q;
    Cache->K = K;
    Cache->V = V;

    Cache->d_attention_score_cache = d_attention_score_cache;
    Cache->d_feedforward_cache_gate = d_feedforward_cache_gate;
    Cache->d_feedforward_cache_up = d_feedforward_cache_up;

    Cache->next_token = next_token;

    return Cache;
}

/* ********************************* Inference Code ********************************* */
void inference(Llama3 *llama3_model, Tensor *X, int *d_tokens, int *h_tokens, CudaCache *Cache) {
    int embed_size = 4096;
    hipMemcpyToSymbol(HIP_SYMBOL(EMBED_SIZE), &embed_size, sizeof(int));

    // Set NUM_TOKENS value in device memory
    h_NUM_TOKENS = h_tokens[0] - 1;
    hipMemcpyToSymbol(HIP_SYMBOL(d_NUM_TOKENS), &h_NUM_TOKENS, sizeof(int));
    free(h_tokens);

    tokens_to_embeddings(X, llama3_model, d_tokens);

    for (int i = 0; i < llama3_model->n_layers; i++) {
        // Pre-attention normalization
        _deviceMemcpy_fp16_tensor(Cache->PN_X, X);
        compute_layer_norm(llama3_model->layers[i]->input_layernorm, X);

        // Attention tensor computation
        compute_qkv_tensors(Cache->Q, Cache->K, Cache->V, llama3_model->layers[i], X);
        exit(1);
        // RoPE scaling
        rope_scaling(Cache->Q, Cache->K);

        // Attention computation
        compute_attention(X, Cache->Q, Cache->K, Cache->V, Cache);

        // Output computation
        compute_output(llama3_model->layers[i], X);

        // Add pre-normalized input
        add_norm(X, Cache->PN_X);

        // Post-attention normalization
        _deviceMemcpy_fp16_tensor(Cache->PN_X, X);
        compute_layer_norm(llama3_model->layers[i]->post_attention_layernorm, X);

        // Feedforward
        compute_feedforward(X, llama3_model->layers[i], Cache);

        // Add pre-normalized input
        add_norm(X, Cache->PN_X);
    }

    compute_layer_norm(llama3_model->norm, X);
    compute_lm_head(llama3_model->lm_head, X, Cache);

    printCudaMemoryInfo();

    return;
}

/* ************************** Convert Tokens to Embeddings ************************** */
void tokens_to_embeddings(Tensor *X, Llama3 *llama3_model, int *d_tokens) {
    // Order threads into blocks
    int total_threads = *(X->mem_len);
    int blocks = (total_threads + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;

    kernel_tokens_to_embeddings<<<blocks, MAX_THREADS_PER_BLOCK>>>(
        X->d_fp16_tensor, d_tokens, llama3_model->embed_tokens->d_fp16_tensor);
    hipDeviceSynchronize();

    return;
}

__global__ void kernel_tokens_to_embeddings(__half *X, int *tokens, __half *Embed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int total_elements = d_NUM_TOKENS * EMBED_SIZE;

    if (idx >= total_elements) return;

    int token_idx = idx / EMBED_SIZE;
    int embed_idx = idx % EMBED_SIZE;

    X[(token_idx * EMBED_SIZE) + embed_idx] =
        Embed[(tokens[token_idx + 1] * EMBED_SIZE) + embed_idx];

    return;
}

/* ******************************* Layer Normalization ******************************* */
// Helpers
Tensor *_create_intermediary_prenorm_tensor_copy() {
    Tensor *Y = (Tensor *)malloc(sizeof(Tensor));

    int *d_ndim;
    int *d_mem_len;
    int *d_shape;
    __half *d_fp16_tensor;

    Y->ndim = (int *)malloc(sizeof(int));
    *(Y->ndim) = 2;

    Y->mem_len = (int *)malloc(sizeof(int));
    *(Y->mem_len) = 4096 * 2048;

    Y->shape = (int *)malloc(sizeof(int) * 2);
    Y->shape[0] = 2048;
    Y->shape[1] = 4096;

    // Allocate CUDA memory
    hipMalloc(&d_ndim, sizeof(int));
    hipMalloc(&d_mem_len, sizeof(int));
    hipMalloc(&d_shape, sizeof(int) * (*(Y->ndim)));
    hipMalloc(&d_fp16_tensor, sizeof(__half) * (*(Y->mem_len)));

    // Copy data to device
    hipMemcpy(d_ndim, Y->ndim, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mem_len, Y->mem_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_shape, Y->shape, sizeof(int) * (*(Y->ndim)), hipMemcpyHostToDevice);

    // Assign device pointers
    Y->d_ndim = d_ndim;
    Y->d_mem_len = d_mem_len;
    Y->d_shape = d_shape;
    Y->d_fp16_tensor = d_fp16_tensor;

    return Y;
}

void _deviceMemcpy_fp16_tensor(Tensor *Y, Tensor *X) {
    hipMemcpy(
        Y->d_fp16_tensor,
        X->d_fp16_tensor,
        sizeof(__half) * (*(X->mem_len)),
        hipMemcpyDeviceToDevice);

    return;
}

// Compute RMS Norm
void compute_layer_norm(Tensor *RMSNorm, Tensor *X) {
    dim3 block(1024);
    dim3 grid(h_NUM_TOKENS);

    kernel_compute_rms_norm<<<grid, block>>>(
        X->d_fp16_tensor, RMSNorm->d_fp16_tensor);
    hipDeviceSynchronize();

    return;
}

__global__ void kernel_compute_rms_norm(__half *X, __half *RMSNorm) {
    __shared__ float shared_mem[1024];

    int token_idx = blockIdx.x;
    int vw_embed_idx = threadIdx.x;

    if (token_idx >= d_NUM_TOKENS) return;
    if (vw_embed_idx >= 1024) return;

    /*
        - Coalesced load into shared memory of 1024 window with vectorized retrieval
        - A 1024 thread block is used to retrieve 4096 elements. Each thread retrieves consecutive
            indicies. Instead of looping and having 4 separate memory access transactions for each
            window retrieval per thread, a singular call loading 4 __half's as 1 uint64_t allows for
            4 indicies to be retreived virtually as one data type.
    */
    c_half4 data = ((c_half4 *)X)[token_idx * 1024 + vw_embed_idx];
    shared_mem[vw_embed_idx] = __half2float(data.x) * __half2float(data.x) +
                               __half2float(data.y) * __half2float(data.y) +
                               __half2float(data.z) * __half2float(data.z) +
                               __half2float(data.w) * __half2float(data.w);
    __syncthreads();

    /*
        - Parallel reduction along y-axis (maximize warp usage without warp divergence)
        - For a 32 x 32 block dimension, the 1st warp will sum with the 16th warp and
            recursively reduce
    */
    for (int offset = 512; offset > 32; offset /= 2) {
        if (vw_embed_idx < offset) {
            shared_mem[vw_embed_idx] += shared_mem[offset + vw_embed_idx];
        }
        __syncthreads();
    }

    /*
        - Parallel reduction for 1 warp (divergent warp behavior) without using shared memory
        - Warp level primitive usage
        - Instead of utilizing shared memory to store intermediate reduction sums, inter-thread
            memory access enables faster reduction
        - For a given warp, the following will still not diverge with 0xffffff mask enabling the
            same instruction for every thread in the warp
        - Offset enables reduction to happen with left most indices lasting the longest. Least
            significant indices still perform addition but add no value to context
    */
    if (vw_embed_idx < 32) {
        __syncthreads();
        float val = shared_mem[vw_embed_idx];
        for (int offset = 16; offset > 0; offset /= 2) {
            val += __shfl_down_sync(0xffffffff, val, offset);
            __syncthreads();
        }
        if (vw_embed_idx == 0) shared_mem[0] = val;
    }

    /*
        - Load rms norm for tensor and perform normalization for 1024 window
        - Similar technique to when loading data from global memory
    */
    float rms = sqrtf((shared_mem[0] / 4096.0f) + 1e-05);
    __syncthreads();
    c_half4 norm_gain = ((c_half4 *)RMSNorm)[vw_embed_idx];

    // Perform RMS calculations and store
    data.x = __float2half(__half2float(data.x) * __half2float(norm_gain.x) / rms);
    data.y = __float2half(__half2float(data.y) * __half2float(norm_gain.y) / rms);
    data.z = __float2half(__half2float(data.z) * __half2float(norm_gain.z) / rms);
    data.w = __float2half(__half2float(data.w) * __half2float(norm_gain.w) / rms);

    ((c_half4 *)X)[token_idx * 1024 + vw_embed_idx] = data;

    return;
}

// Compute addition (skip connection)
void add_norm(Tensor *X, Tensor *PN_X) {
    dim3 block(32, 32, 1);
    dim3 grid(4, h_NUM_TOKENS);

    add_norm<<<grid, block>>>(
        X->d_fp16_tensor, PN_X->d_fp16_tensor);
    hipDeviceSynchronize();

    return;
}

__global__ void add_norm(__half *X, __half *PN_X) {
    int token_idx = blockIdx.y;
    int embed_idx = blockIdx.x * 1024 +
                    threadIdx.y * blockDim.x +
                    threadIdx.x;

    if (token_idx >= d_NUM_TOKENS) return;
    if (embed_idx >= 4096) return;

    int offset = token_idx * 4096 + embed_idx;
    X[offset] = __hadd(X[offset], PN_X[offset]);

    return;
}

/* ***************************** General Matrix Multiplication **************************** */
__global__ void kernel_standard_tiled_gemm(
    __half *O, __half *X, __half *Transform, int m, int n, int k, int TILE_SIZE) {
    /*
        - m represents the independent dimension of the input matrix
        - n represents the independent dimenion of the transformation matrix
        - k represents the common dimension of the 2 matrices
        - Within each kernel, the output is computed as: O = matmul(X, Transform)
        - Transposing the transformation tensor is not required as virtual indexing allows for
            intended navigation along rows and columns of either tensors
        - Order of variables within kernels obey order of computation
    */
    // Kernel start
    //
    extern __shared__ float shared_mem[];
    float *X_shmem = shared_mem;
    float *T_shmem = shared_mem + TILE_SIZE * TILE_SIZE;

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    // Loop over tiles
    float value = 0.0f;
    for (int t = 0; t < (k + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Load tile of X into shared memory
        if (row < m && t * TILE_SIZE + threadIdx.x < k) {
            int X_idx = row * k + t * TILE_SIZE + threadIdx.x;
            X_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = __half2float(X[X_idx]);
        } else {
            X_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = 0.0f;
        }

        // Load tile of Transform into shared memory
        if ((t * TILE_SIZE + threadIdx.y) < k && col < n) {
            int T_idx = col * k + t * TILE_SIZE + threadIdx.y;
            T_shmem[threadIdx.x * TILE_SIZE + threadIdx.y] = __half2float(Transform[T_idx]);
        } else {
            T_shmem[threadIdx.x * TILE_SIZE + threadIdx.y] = 0.0f;
        }
        __syncthreads();

        // Compute partial sums
        for (int i = 0; i < TILE_SIZE; ++i) {
            value += X_shmem[threadIdx.y * TILE_SIZE + i] * T_shmem[i * TILE_SIZE + threadIdx.x];
        }
    }

    // Write the result to global memory
    if (row < m && col < n) {
        int O_idx = row * n + col;
        O[O_idx] = __float2half(value);
    }

    return;
}

/* ***************************** Attention Tensor Computation **************************** */
Tensor *_create_intermediary_attention_tensor(Tensor *Linear) {
    Tensor *Attention_Tensor = (Tensor *)malloc(sizeof(Tensor));

    int *d_ndim;
    int *d_mem_len;
    int *d_shape;
    __half *d_fp16_tensor;

    Attention_Tensor->ndim = (int *)malloc(sizeof(int));
    *(Attention_Tensor->ndim) = 2;

    Attention_Tensor->mem_len = (int *)malloc(sizeof(int));
    *(Attention_Tensor->mem_len) = Linear->shape[0] * 2048;

    Attention_Tensor->shape = (int *)malloc(sizeof(int) * 2);
    Attention_Tensor->shape[0] = 2048;
    Attention_Tensor->shape[1] = Linear->shape[0];

    // Allocate CUDA memory
    hipMalloc(&d_ndim, sizeof(int));
    hipMalloc(&d_mem_len, sizeof(int));
    hipMalloc(&d_shape, sizeof(int) * 2);
    hipMalloc(&d_fp16_tensor, sizeof(__half) * (*(Attention_Tensor->mem_len)));

    // Copy data to device
    hipMemcpy(d_ndim, Attention_Tensor->ndim, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mem_len, Attention_Tensor->mem_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_shape, Attention_Tensor->shape, sizeof(int) * 2, hipMemcpyHostToDevice);

    // Assign device pointers
    Attention_Tensor->d_ndim = d_ndim;
    Attention_Tensor->d_mem_len = d_mem_len;
    Attention_Tensor->d_shape = d_shape;
    Attention_Tensor->d_fp16_tensor = d_fp16_tensor;

    return Attention_Tensor;
}

void compute_qkv_tensors(
    Tensor *Q, Tensor *K, Tensor *V,
    Llama3Layer *L3_Layer, Tensor *X) {
    // Declare common variables
    int TILE_SIZE = 32;
    size_t shared_mem_size = 2 * TILE_SIZE * TILE_SIZE * sizeof(float);
    dim3 block(TILE_SIZE, TILE_SIZE, 1);
    dim3 grid;

    // Query computation
    grid = dim3(
        (L3_Layer->self_attn_q_proj->shape[0] + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE);

    kernel_standard_tiled_gemm<<<grid, block, shared_mem_size>>>(
        Q->d_fp16_tensor, X->d_fp16_tensor, L3_Layer->self_attn_q_proj->d_fp16_tensor,
        h_NUM_TOKENS, L3_Layer->self_attn_q_proj->shape[0], 4096, TILE_SIZE);

    // Key computation
    grid = dim3(
        (L3_Layer->self_attn_k_proj->shape[0] + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE);

    kernel_standard_tiled_gemm<<<grid, block, shared_mem_size>>>(
        K->d_fp16_tensor, X->d_fp16_tensor, L3_Layer->self_attn_k_proj->d_fp16_tensor,
        h_NUM_TOKENS, L3_Layer->self_attn_k_proj->shape[0], 4096, TILE_SIZE);

    // Value computation
    grid = dim3(
        (L3_Layer->self_attn_v_proj->shape[0] + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE);

    kernel_standard_tiled_gemm<<<grid, block, shared_mem_size>>>(
        V->d_fp16_tensor, X->d_fp16_tensor, L3_Layer->self_attn_v_proj->d_fp16_tensor,
        h_NUM_TOKENS, L3_Layer->self_attn_v_proj->shape[0], 4096, TILE_SIZE);
    hipDeviceSynchronize();

    check_embedding<<<1, 1>>>(Q->d_fp16_tensor, 4096);
    hipDeviceSynchronize();

    return;
}

void compute_output(Llama3Layer *L3_Layer, Tensor *X) {
    // Declare common variables
    int TILE_SIZE = 32;
    size_t shared_mem_size = 2 * TILE_SIZE * TILE_SIZE * sizeof(float);
    dim3 block(TILE_SIZE, TILE_SIZE, 1);
    dim3 grid;

    // Output computation
    grid = dim3(
        (L3_Layer->self_attn_o_proj->shape[0] + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE);

    kernel_standard_tiled_gemm<<<grid, block, shared_mem_size>>>(
        X->d_fp16_tensor, X->d_fp16_tensor, L3_Layer->self_attn_o_proj->d_fp16_tensor,
        h_NUM_TOKENS, L3_Layer->self_attn_o_proj->shape[0], 4096, TILE_SIZE);
    hipDeviceSynchronize();

    return;
}

/* ************************* Rotary Positional Embedding (RoPE) ************************* */
void rope_scaling(Tensor *Q, Tensor *K) {
    dim3 block;
    dim3 grid;

    // RoPE on Q
    block = dim3(32, 32, 1);
    grid = dim3(2, h_NUM_TOKENS);
    kernel_rope_scaling<<<grid, block>>>(Q->d_fp16_tensor, 2048);

    // RoPE on K
    block = dim3(16, 16, 1);
    grid = dim3(2, h_NUM_TOKENS);
    kernel_rope_scaling<<<grid, block>>>(K->d_fp16_tensor, 512);

    hipDeviceSynchronize();

    return;
}

__global__ void kernel_rope_scaling(__half *tensor, int transformed_embed_size) {
    /*
        - For Q [tokens, 4096], there are 1024 threads per block with 2 blocks representing one
            transformed Q embedding
        - For K [tokens, 1024], there are 256 threads per block with 2 blocks representing one
            transformed K embedding
        - Window dim gives half the transformed tensor embedding size
        - Window idx gives local index
    */
    int token_idx = blockIdx.y;
    int window_idx = 2 * (blockIdx.x * blockDim.y * blockDim.x +
                          threadIdx.y * blockDim.x +
                          threadIdx.x);

    if (window_idx >= transformed_embed_size) return;
    if (token_idx >= d_NUM_TOKENS) return;

    // Each thread loads 2 __half (each 2 bytes), as one 4 byte value into half2 datatype
    __half2 h2_val = ((const __half2 *)tensor)[window_idx];

    const float scaling_factor = 500000.0f;
    float theta = token_idx / powf(scaling_factor, ((float)window_idx) / ((float)transformed_embed_size));
    float cos_comp = cosf(theta);
    float sin_comp = sinf(theta);

    // Access both values interpreted as 1 and rotate vector pair
    float even = __half2float(__low2half(h2_val));
    float odd = __half2float(__high2half(h2_val));

    float ret_even = (cos_comp * even) - (sin_comp * odd);
    float ret_odd = (sin_comp * even) + (cos_comp * odd);

    // Pack the two __half values into a single __half2
    __half h_ret_even = __float2half(ret_even);
    __half h_ret_odd = __float2half(ret_odd);
    __half2 h2_result = __halves2half2(h_ret_even, h_ret_odd);

    // Store rope encoded data back to tensor
    ((__half2 *)tensor)[window_idx] = h2_result;

    return;
}

/* **************************** Grouped Multi-Query Attention **************************** */
void compute_attention(Tensor *X, Tensor *Q, Tensor *K, Tensor *V, CudaCache *Cache) {
    // Attention score computation
    int TILE_SIZE = 32;
    int nheads = 32;
    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid(
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE,
        nheads);

    size_t shared_mem_size = 2 * TILE_SIZE * TILE_SIZE * sizeof(float);
    kernel_compute_masked_gmq_attention_scores_tiled_matmul<<<grid, block, shared_mem_size>>>(
        Cache->d_attention_score_cache, Q->d_fp16_tensor, K->d_fp16_tensor,
        h_NUM_TOKENS, h_NUM_TOKENS, 128, TILE_SIZE, nheads);
    hipDeviceSynchronize();

    // Masking and softmax
    block = dim3(MAX_THREADS_PER_BLOCK);
    grid = dim3(h_NUM_TOKENS, nheads);

    shared_mem_size = (2048 + 1024) * sizeof(float);
    kernel_masking_softmax<<<grid, block, shared_mem_size>>>(
        Cache->d_attention_score_cache, 1, 1);
    hipDeviceSynchronize();

    // Resolution of attention scores
    block = dim3(TILE_SIZE, TILE_SIZE);
    grid = dim3(
        (128 + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE,
        nheads);

    kernel_compute_resolved_value_from_attention_score_tiled_matmul<<<grid, block, shared_mem_size>>>(
        X->d_fp16_tensor, Cache->d_attention_score_cache, V->d_fp16_tensor,
        h_NUM_TOKENS, h_NUM_TOKENS, 128, nheads, TILE_SIZE);
    hipDeviceSynchronize();

    return;
}

__global__ void kernel_compute_masked_gmq_attention_scores_tiled_matmul(
    float *attention_scores, __half *Q, __half *K,
    int m, int n, int k, int TILE_SIZE, int nheads) {
    /*
        - Each head operates independently of other heads.
        - `m` represents the independent dimension of the Q matrix (number of tokens).
        - `n` represents the independent dimension of the K matrix (number of tokens).
        - `k` represents the common dimension (embedding dimension for each head).
    */

    extern __shared__ float shared_mem[];

    float *Q_shmem = shared_mem;
    float *K_shmem = shared_mem + (TILE_SIZE * TILE_SIZE);

    int head_idx = blockIdx.z;
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    // Loop over tiles
    float value = 0.0f;
    for (int t = 0; t < (k + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        int embedding_idx = t * TILE_SIZE + threadIdx.x;

        // Load tile of Q into shared memory
        if (row < m && embedding_idx < k) {
            int Q_idx = row * nheads * k + head_idx * k + embedding_idx;
            Q_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = __half2float(Q[Q_idx]);
        } else {
            Q_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = 0.0f;
        }

        // Load tile of K into shared memory (transposed)
        if (col < n && embedding_idx < k) {
            int K_idx = (col * nheads * k / 4) + (head_idx * k / 4) + embedding_idx;
            K_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = __half2float(K[K_idx]);
        } else {
            K_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = 0.0f;
        }

        // Compute partial sums
        for (int i = 0; i < TILE_SIZE; ++i) {
            value += Q_shmem[threadIdx.y * TILE_SIZE + i] * K_shmem[i * TILE_SIZE + threadIdx.x];
        }
    }

    // Write the result to shared memory
    if (row < m && col < n) {
        int O_idx = row * nheads * n + head_idx * n + col;
        attention_scores[O_idx] = value / sqrtf((float)k);
    }

    return;
}

__global__ void kernel_masking_softmax(float *attention_scores, int causal_mask, int softmax) {
    extern __shared__ float shared_mem[];

    float *buffer = shared_mem + 2048;

    int token_idx = blockIdx.x;
    int head_idx = blockIdx.y;

    int idx;
    float exp_sum = 0.0f;
    for (int i = 0; i < 2; i++) {
        idx = i * blockDim.x + threadIdx.x;

        if (idx >= d_NUM_TOKENS) {
            shared_mem[idx] = 0.0f;
            continue;
        }

        if (causal_mask) {
            if (idx <= token_idx) {
                shared_mem[idx] = attention_scores[blockIdx.y * 2048 + head_idx * 32 + idx];
            }
        } else {
            shared_mem[idx] = -INFINITY;
        }

        exp_sum += expf(shared_mem[idx]);
    }
    __syncthreads();

    buffer[threadIdx.x] = exp_sum;
    if (softmax) {
        for (int offset = 512; offset > 32; offset /= 2) {
            if (threadIdx.x < offset) {
                buffer[threadIdx.x] += buffer[threadIdx.x + offset];
            }
            __syncthreads();
        }

        if (threadIdx.x < 32) {
            float val = buffer[threadIdx.x];
            for (int offset = 16; offset > 0; offset /= 2) {
                val += __shfl_down_sync(0xffffffff, val, offset);
            }
            if (threadIdx.x == 0) buffer[0] = val;
        }
        __syncthreads();

        float softmax_den = buffer[0];
        for (int i = 0; i < 2; i++) {
            idx = i * blockDim.x + threadIdx.x;
            attention_scores[blockIdx.y * 2048 + head_idx * 32 + idx] = expf(shared_mem[idx]) / softmax_den;
            __syncthreads();
        }
    }

    return;
}

__global__ void kernel_compute_resolved_value_from_attention_score_tiled_matmul(
    __half *output, float *attention_scores, __half *V,
    int m, int k, int d_head, int nheads, int TILE_SIZE) {
    // Kernel start
    //
    extern __shared__ float shared_mem[];
    float *attention_shmem = shared_mem;
    float *V_shmem = shared_mem + TILE_SIZE * TILE_SIZE;

    int head_idx = blockIdx.z;
    int V_head_idx = head_idx / 4;
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float value = 0.0f;

    for (int t = 0; t < (k + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        int k_idx = t * TILE_SIZE + threadIdx.x;  // K dimension index

        // Load attention_scores into shared memory
        if (row < m && k_idx < k) {
            int attn_idx = head_idx * m * k + row * k + k_idx;
            attention_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = attention_scores[attn_idx];
        } else {
            attention_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = 0.0f;
        }

        // Load V into shared memory
        int V_row = k_idx;
        int V_col = col;
        if (V_row < k && V_col < d_head) {
            int V_idx = V_head_idx * k * d_head + V_row * d_head + V_col;
            V_shmem[threadIdx.x * TILE_SIZE + threadIdx.y] = __half2float(V[V_idx]);
        } else {
            V_shmem[threadIdx.x * TILE_SIZE + threadIdx.y] = 0.0f;
        }

        __syncthreads();

        // Compute partial sums
        for (int i = 0; i < TILE_SIZE; ++i) {
            value += attention_shmem[threadIdx.y * TILE_SIZE + i] * V_shmem[i * TILE_SIZE + threadIdx.x];
        }

        __syncthreads();
    }

    // Write the result to the output tensor
    if (row < m && col < d_head) {
        int output_idx = head_idx * m * d_head + row * d_head + col;
        output[output_idx] = __float2half(value);
    }
}

/* ********************************* Feed Forward Network ********************************* */
void compute_feedforward(Tensor *X, Llama3Layer *L3_Layer, CudaCache *Cache) {
    // Declare common variables
    int TILE_SIZE = 32;
    size_t shared_mem_size = 2 * TILE_SIZE * TILE_SIZE * sizeof(float);
    dim3 block(TILE_SIZE, TILE_SIZE, 1);
    dim3 grid;

    // Gate projection computation
    grid = dim3(
        (L3_Layer->mlp_gate_proj->shape[0] + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE);

    kernel_standard_tiled_gemm<<<grid, block, shared_mem_size>>>(
        Cache->d_feedforward_cache_gate, X->d_fp16_tensor, L3_Layer->mlp_gate_proj->d_fp16_tensor,
        h_NUM_TOKENS, L3_Layer->mlp_gate_proj->shape[0], 4096, TILE_SIZE);

    // Up projection computation
    grid = dim3(
        (L3_Layer->mlp_up_proj->shape[0] + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE);

    kernel_standard_tiled_gemm<<<grid, block, shared_mem_size>>>(
        Cache->d_feedforward_cache_up, X->d_fp16_tensor, L3_Layer->mlp_up_proj->d_fp16_tensor,
        h_NUM_TOKENS, L3_Layer->mlp_up_proj->shape[0], 4096, TILE_SIZE);
    hipDeviceSynchronize();

    // Swiglu Activation
    grid = dim3(
        (L3_Layer->mlp_up_proj->shape[0] + 1024 - 1) / 1024,
        h_NUM_TOKENS);

    kernel_compute_swiglu<<<grid, block>>>(
        Cache->d_feedforward_cache_up, Cache->d_feedforward_cache_gate, Cache->d_feedforward_cache_up,
        L3_Layer->mlp_up_proj->shape[0]);
    hipDeviceSynchronize();

    // Final output feedforward output computation
    grid = dim3(
        (L3_Layer->mlp_down_proj->shape[0] + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE);

    kernel_standard_tiled_gemm<<<grid, block, shared_mem_size>>>(
        X->d_fp16_tensor, Cache->d_feedforward_cache_up, L3_Layer->mlp_down_proj->d_fp16_tensor,
        h_NUM_TOKENS, L3_Layer->mlp_down_proj->shape[0], L3_Layer->mlp_up_proj->shape[0], TILE_SIZE);
    hipDeviceSynchronize();

    return;
}

__device__ float sigmoid(float x) {
    return 1 / (1 + expf(-x));
}

__global__ void kernel_compute_swiglu(__half *output, __half *gate, __half *up, int embed_dim) {
    int flat_thread_idx = threadIdx.y * blockDim.x + threadIdx.x;
    int embed_idx = blockIdx.x * blockDim.y * blockDim.x + flat_thread_idx;
    int token_idx = blockIdx.y;

    float gate_val = __half2float(gate[token_idx * embed_dim + embed_idx]);
    float up_val = __half2float(up[token_idx * embed_dim + embed_idx]);

    output[token_idx * embed_dim + embed_idx] = __float2half(
        sigmoid(gate_val) * up_val);

    return;
}

/* ********************************* Language Model Head ********************************* */
void compute_lm_head(Tensor *LM_Head, Tensor *X, CudaCache *Cache) {
    // Declare common variables
    int TILE_SIZE = 32;
    size_t shared_mem_size = 2 * TILE_SIZE * TILE_SIZE * sizeof(float);
    dim3 block(TILE_SIZE, TILE_SIZE, 1);
    dim3 grid;

    // Query computation
    grid = dim3(
        (LM_Head->shape[0] + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE);

    kernel_standard_tiled_gemm<<<grid, block, shared_mem_size>>>(
        Cache->next_token, X->d_fp16_tensor, LM_Head->d_fp16_tensor,
        h_NUM_TOKENS, LM_Head->shape[0], 4096, TILE_SIZE);
    hipDeviceSynchronize();

    check_embedding<<<1, 1>>>(Cache->next_token, 128256);
    hipDeviceSynchronize();

    return;
}
