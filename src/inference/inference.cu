#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "inference.cuh"
#include "llama3/llama3.cuh"

#define CHECK_CUDA_ERROR()                                       \
    {                                                            \
        hipError_t err = hipGetLastError();                    \
        if (err != hipSuccess) {                                \
            printf("CUDA error: %s in file '%s' in line %i\n",   \
                   hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                  \
        }                                                        \
    }

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define MAX(a, b) ((a) > (b) ? (a) : (b))

const int MAX_THREADS_PER_BLOCK = 1024;

__device__ int EMBED_SIZE;

__device__ int d_NUM_TOKENS;
int h_NUM_TOKENS;

/* ************************************ HELPERS ************************************ */
void free_tensor_cuda(Tensor *t) {
    hipFree(t->d_ndim);
    hipFree(t->d_mem_len);
    hipFree(t->d_shape);
    hipFree(t->d_fp16_tensor);

    return;
}

// Print CUDA memory info
void printCudaMemoryInfo() {
    size_t free_memory = 0;
    size_t total_memory = 0;

    // Get the amount of free and total memory on the GPU
    hipError_t err = hipMemGetInfo(&free_memory, &total_memory);

    if (err == hipSuccess) {
        // Convert memory sizes from bytes to megabytes (MB)
        printf("Free GPU Memory: %.2f MB\n", (float)free_memory / (1024 * 1024));
        printf("Total GPU Memory: %.2f MB\n", (float)total_memory / (1024 * 1024));
    } else {
        printf("Failed to get CUDA memory info: %s\n", hipGetErrorString(err));
    }

    return;
}

// Kernel to check and print the embeddings
/*
__global__ void check_embedding(__half *fp16_tensor, int dim) {
    for (int token_idx = 0; token_idx < d_NUM_TOKENS; token_idx++) {
        printf("Token %d embeddings:\n", token_idx);
        for (int i = 0; i < dim; i++) {
            printf("%f, ", __half2float(fp16_tensor[token_idx * dim + i]));
        }
        printf("\n");
        printf("\n\n");
    }

    return;
}
*/
__global__ void check_embedding(__half *fp16_tensor, int dim) {
    for (int token_idx = 0; token_idx < d_NUM_TOKENS; token_idx++) {
        printf("Token %d embeddings:\n", token_idx);
        int max = 0;
        float curr_max = 0.0f;
        for (int i = 0; i < dim; i++) {
            float embedding = __half2float(fp16_tensor[token_idx * dim + i]);

            if (embedding > curr_max) {
                curr_max = embedding;
                max = i;
            }
        }
        printf("%d\n", max);
        printf("\n\n");
    }

    return;
}

/* ************************************* Cache ************************************* */
// Allocate global mem cache on device
void *create_gmemcache(size_t mem_len, size_t type_size) {
    void *d_gcache;

    hipMalloc((void **)&d_gcache, mem_len * type_size);

    return d_gcache;
}

CudaCache *init_cache(Llama3 *llama3_model) {
    // Ahead Of Time memory allocations
    // Allocate once, use everywhere
    CudaCache *Cache = (CudaCache *)malloc(sizeof(CudaCache));

    // Allocate Memory --------------------------------------------------------
    Tensor *PN_X = _create_intermediary_prenorm_tensor_copy();

    Tensor *Q = _create_intermediary_attention_tensor(llama3_model->layers[0]->self_attn_q_proj);
    Tensor *K = _create_intermediary_attention_tensor(llama3_model->layers[0]->self_attn_k_proj);
    Tensor *V = _create_intermediary_attention_tensor(llama3_model->layers[0]->self_attn_v_proj);

    float *d_attention_score_cache = (float *)create_gmemcache(32 * 2048 * 2048, sizeof(float));

    __half *d_feedforward_cache_gate = (__half *)create_gmemcache(2048 * 14336, sizeof(__half));
    __half *d_feedforward_cache_up = (__half *)create_gmemcache(2048 * 14336, sizeof(__half));

    __half *next_token = (__half *)create_gmemcache(128256 * 2048, sizeof(__half));

    // Save pointers to Struct --------------------------------------------------------
    Cache->PN_X = PN_X;

    Cache->Q = Q;
    Cache->K = K;
    Cache->V = V;

    Cache->d_attention_score_cache = d_attention_score_cache;
    Cache->d_feedforward_cache_gate = d_feedforward_cache_gate;
    Cache->d_feedforward_cache_up = d_feedforward_cache_up;

    Cache->next_token = next_token;

    return Cache;
}

/* ********************************* Inference Code ********************************* */
void inference(Llama3 *llama3_model, Tensor *X, int *d_tokens, int *h_tokens, CudaCache *Cache) {
    int embed_size = 4096;
    hipMemcpyToSymbol(HIP_SYMBOL(EMBED_SIZE), &embed_size, sizeof(int));

    // Set NUM_TOKENS value in device memory
    h_NUM_TOKENS = h_tokens[0] - 1;
    hipMemcpyToSymbol(HIP_SYMBOL(d_NUM_TOKENS), &h_NUM_TOKENS, sizeof(int));

    hipEvent_t start, stop;
    float milliseconds;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    printf("------ Inference Timing Log ------\n");

    // Measure tokens_to_embeddings
    hipEventRecord(start, 0);
    tokens_to_embeddings(X, llama3_model, d_tokens);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Function: tokens_to_embeddings | Time: %8.2f ms\n", milliseconds);

    for (int i = 0; i < llama3_model->n_layers; i++) {
        printf("\n--- Layer %d ---\n", i);

        // Pre-attention normalization
        hipEventRecord(start, 0);
        _deviceMemcpy_fp16_tensor(Cache->PN_X, X);
        compute_layer_norm(llama3_model->layers[i]->input_layernorm, X);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Function: Pre-attention normalization       | Time: %8.2f ms\n", milliseconds);

        // Attention tensor computation
        hipEventRecord(start, 0);
        compute_qkv_tensors(Cache->Q, Cache->K, Cache->V, llama3_model->layers[i], X);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Function: compute_qkv_tensors                | Time: %8.2f ms\n", milliseconds);

        // RoPE scaling
        hipEventRecord(start, 0);
        rope_scaling(Cache->Q, Cache->K);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Function: rope_scaling                       | Time: %8.2f ms\n", milliseconds);

        // Attention computation
        hipEventRecord(start, 0);
        compute_attention(X, Cache->Q, Cache->K, Cache->V, Cache);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Function: compute_attention                  | Time: %8.2f ms\n", milliseconds);

        // Output computation
        hipEventRecord(start, 0);
        compute_output(llama3_model->layers[i], X, Cache);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Function: compute_output                     | Time: %8.2f ms\n", milliseconds);

        // Add pre-normalized input
        hipEventRecord(start, 0);
        add_norm(X, Cache->PN_X);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Function: add_norm                           | Time: %8.2f ms\n", milliseconds);

        // Post-attention normalization
        hipEventRecord(start, 0);
        _deviceMemcpy_fp16_tensor(Cache->PN_X, X);
        compute_layer_norm(llama3_model->layers[i]->post_attention_layernorm, X);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Function: Post-attention normalization       | Time: %8.2f ms\n", milliseconds);

        // Feedforward
        hipEventRecord(start, 0);
        compute_feedforward(X, llama3_model->layers[i], Cache);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Function: compute_feedforward                | Time: %8.2f ms\n", milliseconds);

        // Add pre-normalized input after feedforward
        hipEventRecord(start, 0);
        add_norm(X, Cache->PN_X);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Function: add_norm after feedforward         | Time: %8.2f ms\n", milliseconds);
    }

    // Final layer normalization
    hipEventRecord(start, 0);
    compute_layer_norm(llama3_model->norm, X);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("\nFunction: Final layer normalization          | Time: %8.2f ms\n", milliseconds);

    // Language model head computation
    hipEventRecord(start, 0);
    compute_lm_head(llama3_model->lm_head, X, Cache);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Function: compute_lm_head                    | Time: %8.2f ms\n", milliseconds);

    printf("----------------------------------------------\n");

    // Cleanup CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA_ERROR();
    printCudaMemoryInfo();
}

/* ************************** Convert Tokens to Embeddings ************************** */
void tokens_to_embeddings(Tensor *X, Llama3 *llama3_model, int *d_tokens) {
    // Order threads into blocks
    int total_threads = *(X->mem_len);
    int blocks = (total_threads + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;

    kernel_tokens_to_embeddings<<<blocks, MAX_THREADS_PER_BLOCK>>>(
        X->d_fp16_tensor, d_tokens, llama3_model->embed_tokens->d_fp16_tensor,
        h_NUM_TOKENS);
    hipDeviceSynchronize();

    return;
}

__global__ void kernel_tokens_to_embeddings(__half *X, int *tokens, __half *Embed, int num_tokens) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int total_elements = num_tokens * 4096;

    if (idx >= total_elements) return;

    int token_idx = idx / 4096;
    int embed_idx = idx % 4096;

    X[(token_idx * 4096) + embed_idx] =
        Embed[(tokens[token_idx + 1] * 4096) + embed_idx];

    return;
}

/* ******************************* Layer Normalization ******************************* */
// Helpers
Tensor *_create_intermediary_prenorm_tensor_copy() {
    Tensor *Y = (Tensor *)malloc(sizeof(Tensor));

    int *d_ndim;
    int *d_mem_len;
    int *d_shape;
    __half *d_fp16_tensor;

    Y->ndim = (int *)malloc(sizeof(int));
    *(Y->ndim) = 2;

    Y->mem_len = (int *)malloc(sizeof(int));
    *(Y->mem_len) = 2048 * 4096;

    Y->shape = (int *)malloc(sizeof(int) * 2);
    Y->shape[0] = 2048;
    Y->shape[1] = 4096;

    // Allocate CUDA memory
    hipMalloc((void **)&d_ndim, sizeof(int));
    hipMalloc((void **)&d_mem_len, sizeof(int));
    hipMalloc((void **)&d_shape, sizeof(int) * (*(Y->ndim)));
    hipMalloc((void **)&d_fp16_tensor, sizeof(__half) * (*(Y->mem_len)));

    // Copy data to device
    hipMemcpy(d_ndim, Y->ndim, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mem_len, Y->mem_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_shape, Y->shape, sizeof(int) * (*(Y->ndim)), hipMemcpyHostToDevice);
    hipMemset(d_fp16_tensor, __float2half(0.0f), sizeof(__half) * (*(Y->mem_len)));

    // Assign device pointers
    Y->d_ndim = d_ndim;
    Y->d_mem_len = d_mem_len;
    Y->d_shape = d_shape;
    Y->d_fp16_tensor = d_fp16_tensor;

    return Y;
}

void _deviceMemcpy_fp16_tensor(Tensor *Y, Tensor *X) {
    if (*(X->mem_len) > *(Y->mem_len)) {
        printf("X and Y are not alike Tensors: (_deviceMemcpy_fp16_tensor)");
        exit(1);
    };

    hipMemset(Y->d_fp16_tensor, __float2half(0.0f), sizeof(__half) * (*(X->mem_len)));
    hipMemcpy(
        Y->d_fp16_tensor,
        X->d_fp16_tensor,
        sizeof(__half) * (*(X->mem_len)),
        hipMemcpyDeviceToDevice);

    return;
}

// Compute RMS Norm
void compute_layer_norm(Tensor *RMSNorm, Tensor *X) {
    dim3 block(32, 32);
    dim3 grid(h_NUM_TOKENS);

    kernel_compute_rms_norm<<<grid, block>>>(
        X->d_fp16_tensor, RMSNorm->d_fp16_tensor, h_NUM_TOKENS);
    hipDeviceSynchronize();

    return;
}

__global__ void kernel_compute_rms_norm(__half *X, __half *RMSNorm, int num_tokens) {
    __shared__ float shared_mem[1024];

    int token_idx = blockIdx.x;
    int vw_embed_idx = threadIdx.y * blockDim.x + threadIdx.x;

    if (token_idx >= num_tokens) return;
    if (vw_embed_idx >= 1024) return;

    /*
        - Coalesced load into shared memory of 1024 window with vectorized retrieval
        - A 1024 thread block is used to retrieve 4096 elements. Each thread retrieves consecutive
            indicies. Instead of looping and having 4 separate memory access transactions for each
            window retrieval per thread, a singular call loading 4 __half's as 1 uint64_t allows for
            4 indicies to be retrieved virtually as one data type.
    */
    c_half4 data = ((c_half4 *)X)[token_idx * 1024 + vw_embed_idx];
    shared_mem[vw_embed_idx] = __half2float(data.x) * __half2float(data.x) +
                               __half2float(data.y) * __half2float(data.y) +
                               __half2float(data.z) * __half2float(data.z) +
                               __half2float(data.w) * __half2float(data.w);
    __syncthreads();

    /*
        - Parallel reduction along y-axis (maximize warp usage without warp divergence)
        - For a 32 x 32 block dimension, the 1st warp will sum with the 16th warp and
            recursively reduce
    */
    for (int offset = 512; offset >= 32; offset /= 2) {
        if (vw_embed_idx < offset) {
            shared_mem[vw_embed_idx] += shared_mem[offset + vw_embed_idx];
        }
        __syncthreads();
    }

    /*
        - Parallel reduction for 1 warp (divergent warp behavior) without using shared memory
        - Warp level primitive usage
        - Instead of utilizing shared memory to store intermediate reduction sums, inter-thread
            memory access enables faster reduction
        - For a given warp, the following will still not diverge with 0xffffff mask enabling the
            same instruction for every thread in the warp
        - Offset enables reduction to happen with left most indices lasting the longest. Least
            significant indices still perform addition but add no value to context
    */
    if (vw_embed_idx < 32) {
        float val = shared_mem[vw_embed_idx];
        for (int offset = 16; offset > 0; offset /= 2) {
            val += __shfl_down_sync(0xffffffff, val, offset);
        }
        if (vw_embed_idx == 0) shared_mem[0] = val;
    }
    __syncthreads();

    /*
        - Load rms norm for tensor and perform normalization for 1024 window
        - Similar technique to when loading data from global memory
    */
    float rms = sqrtf(1e-5 + (shared_mem[0] / 4096.0f));
    c_half4 norm_gain = ((c_half4 *)RMSNorm)[vw_embed_idx];

    // Perform RMS calculations and store
    data.x = __float2half(__half2float(data.x) * __half2float(norm_gain.x) / rms);
    data.y = __float2half(__half2float(data.y) * __half2float(norm_gain.y) / rms);
    data.z = __float2half(__half2float(data.z) * __half2float(norm_gain.z) / rms);
    data.w = __float2half(__half2float(data.w) * __half2float(norm_gain.w) / rms);

    ((c_half4 *)X)[token_idx * 1024 + vw_embed_idx] = data;

    return;
}

// Compute addition (skip connection)
void add_norm(Tensor *X, Tensor *PN_X) {
    dim3 block(1024);
    dim3 grid(4, h_NUM_TOKENS);

    add_norm<<<grid, block>>>(
        X->d_fp16_tensor, PN_X->d_fp16_tensor, h_NUM_TOKENS);
    hipDeviceSynchronize();

    return;
}

__global__ void add_norm(__half *X, __half *PN_X, int num_tokens) {
    int token_idx = blockIdx.y;
    int embed_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (token_idx >= num_tokens) return;
    if (embed_idx >= 4096) return;

    int offset = token_idx * 4096 + embed_idx;
    float a = __half2float(X[offset]);
    float b = __half2float(PN_X[offset]);
    X[offset] = __float2half(a + b);

    return;
}

/* ***************************** General Matrix Multiplication **************************** */
__global__ void kernel_standard_tiled_gemm(
    __half *O, __half *X, __half *Transform, int m, int n, int k, int TILE_SIZE) {
    /*
        - m represents the independent dimension of the input matrix
        - n represents the independent dimenion of the transformation matrix
        - k represents the common dimension of the 2 matrices
        - Within each kernel, the output is computed as: O = matmul(X, Transform)
        - Transposing the transformation tensor is not required as virtual indexing allows
          for intended navigation along rows and columns of either tensors
        - Order of variables within kernels obey order of computation
    */
    // Kernel start
    //
    extern __shared__ float shared_mem[];
    float *X_shmem = shared_mem;
    float *T_shmem = shared_mem + TILE_SIZE * TILE_SIZE;

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    // Loop over tiles
    float value = 0.0f;
    for (int t = 0; t < ((k + TILE_SIZE - 1) / TILE_SIZE); ++t) {
        // Load tile of X into shared memory
        if (row < m && (t * TILE_SIZE + threadIdx.x) < k) {
            int X_idx = row * k + t * TILE_SIZE + threadIdx.x;
            X_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = __half2float(X[X_idx]);
        } else {
            X_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = 0.0f;
        }

        // Load tile of Transform into shared memory
        if (col < n && (t * TILE_SIZE + threadIdx.y) < k) {
            int T_idx = col * k + t * TILE_SIZE + threadIdx.y;
            T_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = __half2float(Transform[T_idx]);
        } else {
            T_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = 0.0f;
        }
        __syncthreads();

        // Compute partial sums
        for (int i = 0; i < TILE_SIZE; ++i) {
            value += X_shmem[threadIdx.y * TILE_SIZE + i] * T_shmem[i * TILE_SIZE + threadIdx.x];
        }
        __syncthreads();
    }

    // Write the result to global memory
    if (row < m && col < n) {
        O[row * n + col] = __float2half(value);
    }

    return;
}

/* ***************************** Attention Tensor Computation **************************** */
Tensor *_create_intermediary_attention_tensor(Tensor *Linear) {
    Tensor *Attention_Tensor = (Tensor *)malloc(sizeof(Tensor));

    int *d_ndim;
    int *d_mem_len;
    int *d_shape;
    __half *d_fp16_tensor;

    Attention_Tensor->ndim = (int *)malloc(sizeof(int));
    *(Attention_Tensor->ndim) = 2;

    Attention_Tensor->mem_len = (int *)malloc(sizeof(int));
    *(Attention_Tensor->mem_len) = 2048 * Linear->shape[0];

    Attention_Tensor->shape = (int *)malloc(sizeof(int) * 2);
    Attention_Tensor->shape[0] = 2048;
    Attention_Tensor->shape[1] = Linear->shape[0];

    // Allocate CUDA memory
    hipMalloc((void **)&d_ndim, sizeof(int));
    hipMalloc((void **)&d_mem_len, sizeof(int));
    hipMalloc((void **)&d_shape, sizeof(int) * 2);
    hipMalloc((void **)&d_fp16_tensor, sizeof(__half) * (*(Attention_Tensor->mem_len)));

    // Copy data to device
    hipMemcpy(d_ndim, Attention_Tensor->ndim, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mem_len, Attention_Tensor->mem_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_shape, Attention_Tensor->shape, sizeof(int) * 2, hipMemcpyHostToDevice);
    hipMemset(d_fp16_tensor, __float2half(0.0f), sizeof(__half) * (*(Attention_Tensor->mem_len)));

    // Assign device pointers
    Attention_Tensor->d_ndim = d_ndim;
    Attention_Tensor->d_mem_len = d_mem_len;
    Attention_Tensor->d_shape = d_shape;
    Attention_Tensor->d_fp16_tensor = d_fp16_tensor;

    return Attention_Tensor;
}

void compute_qkv_tensors(
    Tensor *Q, Tensor *K, Tensor *V,
    Llama3Layer *L3_Layer, Tensor *X) {
    // Declare common variables
    int TILE_SIZE = 32;
    size_t shared_mem_size = 2 * TILE_SIZE * TILE_SIZE * sizeof(float);
    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid;

    // Query computation
    grid = dim3(
        (L3_Layer->self_attn_q_proj->shape[0] + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE);

    kernel_standard_tiled_gemm<<<grid, block, shared_mem_size>>>(
        Q->d_fp16_tensor, X->d_fp16_tensor, L3_Layer->self_attn_q_proj->d_fp16_tensor,
        h_NUM_TOKENS, L3_Layer->self_attn_q_proj->shape[0], 4096, TILE_SIZE);

    // Key computation
    grid = dim3(
        (L3_Layer->self_attn_k_proj->shape[0] + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE);

    kernel_standard_tiled_gemm<<<grid, block, shared_mem_size>>>(
        K->d_fp16_tensor, X->d_fp16_tensor, L3_Layer->self_attn_k_proj->d_fp16_tensor,
        h_NUM_TOKENS, L3_Layer->self_attn_k_proj->shape[0], 4096, TILE_SIZE);

    // Value computation
    grid = dim3(
        (L3_Layer->self_attn_v_proj->shape[0] + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE);

    kernel_standard_tiled_gemm<<<grid, block, shared_mem_size>>>(
        V->d_fp16_tensor, X->d_fp16_tensor, L3_Layer->self_attn_v_proj->d_fp16_tensor,
        h_NUM_TOKENS, L3_Layer->self_attn_v_proj->shape[0], 4096, TILE_SIZE);
    hipDeviceSynchronize();

    return;
}

void compute_output(Llama3Layer *L3_Layer, Tensor *X, CudaCache *Cache) {
    // Declare common variables
    int TILE_SIZE = 32;
    size_t shared_mem_size = 2 * TILE_SIZE * TILE_SIZE * sizeof(float);
    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid;

    // Output computation
    grid = dim3(
        (L3_Layer->self_attn_o_proj->shape[0] + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE);
    _deviceMemcpy_fp16_tensor(Cache->Q, X);

    kernel_standard_tiled_gemm<<<grid, block, shared_mem_size>>>(
        X->d_fp16_tensor, Cache->Q->d_fp16_tensor, L3_Layer->self_attn_o_proj->d_fp16_tensor,
        h_NUM_TOKENS, L3_Layer->self_attn_o_proj->shape[0], 4096, TILE_SIZE);
    hipDeviceSynchronize();

    return;
}

/* ************************* Rotary Positional Embedding (RoPE) ************************* */
void rope_scaling(Tensor *Q, Tensor *K) {
    dim3 block;
    dim3 grid;

    // RoPE on Q
    block = dim3(1024);
    grid = dim3(2, h_NUM_TOKENS);
    kernel_rope_scaling<<<grid, block>>>(Q->d_fp16_tensor, 2048, h_NUM_TOKENS);

    // RoPE on K
    block = dim3(256);
    grid = dim3(2, h_NUM_TOKENS);
    kernel_rope_scaling<<<grid, block>>>(K->d_fp16_tensor, 512, h_NUM_TOKENS);
    hipDeviceSynchronize();

    return;
}

__global__ void kernel_rope_scaling(__half *tensor, int transformed_embed_size, int num_tokens) {
    /*
        - For Q [tokens, 4096], there are 1024 threads per block with 2 blocks representing one
            transformed Q embedding
        - For K [tokens, 1024], there are 256 threads per block with 2 blocks representing one
            transformed K embedding
        - Window dim gives half the transformed tensor embedding size
        - Window idx gives local index
    */
    int token_idx = blockIdx.y;
    int embed_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (embed_idx >= transformed_embed_size) return;
    if (token_idx >= num_tokens) return;

    // Each thread loads 2 __half (each 2 bytes), as one 4 byte value into half2 datatype
    __half2 h2_val = ((const __half2 *)tensor)[token_idx * transformed_embed_size + embed_idx];

    const float scaling_factor = 500000.0f;
    float theta = (token_idx) / powf(scaling_factor, (embed_idx / transformed_embed_size));
    float cos_comp = cosf(theta);
    float sin_comp = sinf(theta);

    // Access both values interpreted as 1 and rotate vector pair
    float even = __half2float(__low2half(h2_val));
    float odd = __half2float(__high2half(h2_val));

    float ret_even = (cos_comp * even) - (sin_comp * odd);
    float ret_odd = (sin_comp * even) + (cos_comp * odd);

    // Pack the two __half values into a single __half2
    __half h_ret_even = __float2half(ret_even);
    __half h_ret_odd = __float2half(ret_odd);
    __half2 h2_result = __halves2half2(h_ret_even, h_ret_odd);

    __syncthreads();

    // Store rope encoded data back to tensor
    ((__half2 *)tensor)[token_idx * transformed_embed_size + embed_idx] = h2_result;

    return;
}

/* **************************** Grouped Multi-Query Attention **************************** */
void compute_attention(Tensor *X, Tensor *Q, Tensor *K, Tensor *V, CudaCache *Cache) {
    // Attention score computation
    int TILE_SIZE = 32;
    int nheads = 32;
    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid(
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE,
        nheads);

    size_t shared_mem = 2 * TILE_SIZE * TILE_SIZE * sizeof(float);
    kernel_compute_masked_gmq_attention_scores_tiled_matmul<<<grid, block, shared_mem>>>(
        Cache->d_attention_score_cache, Q->d_fp16_tensor, K->d_fp16_tensor,
        h_NUM_TOKENS, h_NUM_TOKENS, 128, TILE_SIZE, nheads);
    hipDeviceSynchronize();

    block = dim3(1024);
    grid = dim3(h_NUM_TOKENS, nheads);

    shared_mem = (2048 + 1024) * sizeof(float);
    kernel_masking_softmax<<<grid, block, shared_mem>>>(
        Cache->d_attention_score_cache, h_NUM_TOKENS);
    hipDeviceSynchronize();

    block = dim3(TILE_SIZE, TILE_SIZE);
    grid = dim3(
        (128 + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE,
        nheads);

    shared_mem = 2 * TILE_SIZE * TILE_SIZE * sizeof(float);
    kernel_compute_resolved_value_from_attention_score_tiled_matmul<<<grid, block, shared_mem>>>(
        X->d_fp16_tensor, Cache->d_attention_score_cache, V->d_fp16_tensor,
        h_NUM_TOKENS, 128, h_NUM_TOKENS, nheads, TILE_SIZE);
    hipDeviceSynchronize();

    return;
}

__global__ void kernel_compute_masked_gmq_attention_scores_tiled_matmul(
    float *attention_scores, __half *Q, __half *K,
    int m, int n, int k, int TILE_SIZE, int nheads) {
    /*
        - Each head operates independently of other heads.
        - `m`: Number of tokens (rows of Q).
        - `n`: Number of tokens (columns of K).
        - `k`: Head dimension (common dimension).
        - `nheads`: Number of attention heads.
    */

    extern __shared__ float shared_mem[];
    float *Q_shmem = shared_mem;
    float *K_shmem = shared_mem + (TILE_SIZE * TILE_SIZE);

    int q_head_idx = blockIdx.z;
    int kv_head_idx = q_head_idx / 4;
    int kv_heads = nheads / 4;

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float value = 0.0f;
    for (int t = 0; t < (k + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        if (row < m && (t * TILE_SIZE + threadIdx.x) < k) {
            int Q_idx = row * (nheads * k) + (q_head_idx * k) + t * TILE_SIZE + threadIdx.x;
            Q_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = __half2float(Q[Q_idx]);
        } else {
            Q_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = 0.0f;
        }

        if (col < n && (t * TILE_SIZE + threadIdx.y) < k) {
            int K_idx = col * (kv_heads * k) + (kv_head_idx * k) + t * TILE_SIZE + threadIdx.y;
            K_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = __half2float(K[K_idx]);
        } else {
            K_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = 0.0f;
        }
        __syncthreads();

        // Compute partial sums
        for (int i = 0; i < TILE_SIZE; i++) {
            value += Q_shmem[threadIdx.y * TILE_SIZE + i] * K_shmem[i * TILE_SIZE + threadIdx.x];
        }
        __syncthreads();
    }

    // Write result to memory
    if (row < m && col < n) {
        attention_scores[q_head_idx * m * n + row * n + col] = value / sqrtf(k);
    }

    return;
}

__global__ void kernel_masking_softmax(float *attention_scores, int num_tokens) {
    extern __shared__ float shared_mem[];
    float *vec = shared_mem;
    float *buffer = shared_mem + 2048;

    int token_idx_y = blockIdx.x;
    int head_idx = blockIdx.y;

    if (token_idx_y >= num_tokens) return;
    if (head_idx >= 32) return;

    int token_idx_x;
    float exp_sum = 0.0f;

    // Load relevant attention scores and apply masking
    for (int i = 0; i < (num_tokens + blockDim.x - 1) / blockDim.x; i++) {
        token_idx_x = i * blockDim.x + threadIdx.x;

        if (token_idx_x < num_tokens) {
            if (token_idx_x <= token_idx_y) {
                vec[token_idx_x] = attention_scores[(head_idx * num_tokens * num_tokens) + (token_idx_y * num_tokens) + token_idx_x];
                exp_sum += expf(vec[token_idx_x]);
            } else {
                vec[token_idx_x] = 0.0f;
            }
        } else {
            vec[token_idx_x] = 0.0f;
        }
        __syncthreads();
    }

    // Reduction to compute softmax denominator
    buffer[threadIdx.x] = exp_sum;
    __syncthreads();

    for (int offset = 512; offset > 0; offset /= 2) {
        if (threadIdx.x < offset) {
            buffer[threadIdx.x] += buffer[threadIdx.x + offset];
        }
        __syncthreads();
    }

    float softmax_den = buffer[0];
    __syncthreads();

    // Compute softmax
    for (int i = 0; i < (num_tokens + blockDim.x - 1) / blockDim.x; i++) {
        token_idx_x = i * blockDim.x + threadIdx.x;
        if (token_idx_x < num_tokens) {
            if (token_idx_x <= token_idx_y) {
                attention_scores[(head_idx * num_tokens * num_tokens) + (token_idx_y * num_tokens) + token_idx_x] = expf(vec[token_idx_x]) / softmax_den;
            } else {
                attention_scores[(head_idx * num_tokens * num_tokens) + (token_idx_y * num_tokens) + token_idx_x] = 0.0f;
            }
        }
        __syncthreads();
    }

    return;
}

__global__ void kernel_compute_resolved_value_from_attention_score_tiled_matmul(
    __half *output, float *attention_scores, __half *V,
    int m, int n, int k, int nheads, int TILE_SIZE) {
    /*
        - Each head operates independently of other heads.
        - `m`: Number of tokens (rows of attention scores).
        - `n`: Head dimension
        - `k`: Number of tokens (common dimension).
        - `TILE_SIZE`: Tile size for shared memory.
    */

    extern __shared__ float shared_mem[];
    float *attention_shmem = shared_mem;
    float *V_shmem = shared_mem + TILE_SIZE * TILE_SIZE;

    int q_head_idx = blockIdx.z;
    int kv_head_idx = q_head_idx / 4;
    int kv_heads = nheads / 4;

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float value = 0.0f;
    for (int t = 0; t < (k + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        // Load attention_scores into shared memory
        if (row < m && (t * TILE_SIZE + threadIdx.x) < k) {
            int attn_idx = q_head_idx * m * k + row * k + (t * TILE_SIZE + threadIdx.x);
            attention_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = attention_scores[attn_idx];
        } else {
            attention_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = 0.0f;
        }

        // Load V into shared memory
        if (col < n && (t * TILE_SIZE + threadIdx.y) < k) {
            int V_idx = (t * TILE_SIZE * n * kv_heads) + (threadIdx.y * n * kv_heads) + kv_head_idx * n + col;
            V_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = __half2float(V[V_idx]);
        } else {
            V_shmem[threadIdx.y * TILE_SIZE + threadIdx.x] = 0.0f;
        }
        __syncthreads();

        // Compute partial sums
        for (int i = 0; i < TILE_SIZE; i++) {
            if ((t * TILE_SIZE + i) < k) {
                value += attention_shmem[threadIdx.y * TILE_SIZE + i] * V_shmem[i * TILE_SIZE + threadIdx.x];
            }
        }
        __syncthreads();
    }

    // Write the result to the output tensor
    if (row < m && col < n) {
        int output_idx = row * nheads * n + q_head_idx * n + col;
        output[output_idx] = __float2half(value);
    }
}

/* ********************************* Feed Forward Network ********************************* */
void compute_feedforward(Tensor *X, Llama3Layer *L3_Layer, CudaCache *Cache) {
    // Declare common variables
    int TILE_SIZE = 32;
    size_t shared_mem_size = 2 * TILE_SIZE * TILE_SIZE * sizeof(float);
    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid;

    // Gate projection computation
    grid = dim3(
        (L3_Layer->mlp_gate_proj->shape[0] + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE);

    kernel_standard_tiled_gemm<<<grid, block, shared_mem_size>>>(
        Cache->d_feedforward_cache_gate, X->d_fp16_tensor, L3_Layer->mlp_gate_proj->d_fp16_tensor,
        h_NUM_TOKENS, L3_Layer->mlp_gate_proj->shape[0], 4096, TILE_SIZE);
    hipDeviceSynchronize();

    // Up projection computation
    grid = dim3(
        (L3_Layer->mlp_up_proj->shape[0] + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE);

    kernel_standard_tiled_gemm<<<grid, block, shared_mem_size>>>(
        Cache->d_feedforward_cache_up, X->d_fp16_tensor, L3_Layer->mlp_up_proj->d_fp16_tensor,
        h_NUM_TOKENS, L3_Layer->mlp_up_proj->shape[0], 4096, TILE_SIZE);
    hipDeviceSynchronize();

    // Swiglu Activation
    grid = dim3(
        (L3_Layer->mlp_up_proj->shape[0] + 1024 - 1) / 1024,
        h_NUM_TOKENS);

    kernel_compute_swiglu<<<grid, 1024>>>(
        Cache->d_feedforward_cache_up, Cache->d_feedforward_cache_gate, Cache->d_feedforward_cache_up,
        L3_Layer->mlp_up_proj->shape[0], h_NUM_TOKENS);
    hipDeviceSynchronize();

    // Final output feedforward output computation
    grid = dim3(
        (L3_Layer->mlp_down_proj->shape[0] + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE);

    kernel_standard_tiled_gemm<<<grid, block, shared_mem_size>>>(
        X->d_fp16_tensor, Cache->d_feedforward_cache_up, L3_Layer->mlp_down_proj->d_fp16_tensor,
        h_NUM_TOKENS, L3_Layer->mlp_down_proj->shape[0], L3_Layer->mlp_down_proj->shape[1], TILE_SIZE);
    hipDeviceSynchronize();

    return;
}

__device__ float SiLU(float x) {
    return x / (1 + expf(x * -1.0f));
}

__global__ void kernel_compute_swiglu(
    __half *output, __half *gate, __half *up,
    int embed_dim, int num_tokens) {
    // Kernel start
    //
    int embed_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int token_idx = blockIdx.y;

    if (token_idx >= num_tokens) return;
    if (embed_idx >= embed_dim) return;

    int index = token_idx * embed_dim + embed_idx;

    float gate_val = __half2float(gate[index]);
    float up_val = __half2float(up[index]);

    output[index] = __float2half(SiLU(gate_val) * up_val);

    return;
}

/* ********************************* Language Model Head ********************************* */
void compute_lm_head(Tensor *LM_Head, Tensor *X, CudaCache *Cache) {
    // Declare common variables
    int TILE_SIZE = 32;
    size_t shared_mem_size = 2 * TILE_SIZE * TILE_SIZE * sizeof(float);
    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid;

    // Query computation
    grid = dim3(
        (LM_Head->shape[0] + TILE_SIZE - 1) / TILE_SIZE,
        (h_NUM_TOKENS + TILE_SIZE - 1) / TILE_SIZE);

    kernel_standard_tiled_gemm<<<grid, block, shared_mem_size>>>(
        Cache->next_token, X->d_fp16_tensor, LM_Head->d_fp16_tensor,
        h_NUM_TOKENS, LM_Head->shape[0], 4096, TILE_SIZE);
    hipDeviceSynchronize();

    // check_embedding<<<1, 1>>>(Cache->next_token, 128256);
    // hipDeviceSynchronize();

    return;
}
