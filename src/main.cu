#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "inference/inference.cuh"
#include "llama3/llama3.cuh"
#include "safetensor/safetensor.cuh"
#include "tokenizer/tokenizer.cuh"

#define WARN "\033[1;33m"
#define GREEN "\033[1;32m"
#define GREY "\033[2m"
#define RESET "\033[0m"

#define CLEAR_TERMINAL() system("clear")

const int MODEL_NUM_LAYERS = 32;

int main() {
    // Initialize the Llama3 model
    Llama3 *llama3_model = init_llama3(MODEL_NUM_LAYERS);

    if (llama3_model == NULL) {
        printf("An error occurred while allocating memory for the Llama3 struct\n");
        exit(1);
    } else {
        printf("Model has been allocated with %d layers\n", llama3_model->n_layers);
    }

    // Load the safetensor weights into the model
    load_safetensor_weights(llama3_model, "model_weights/model-00001-of-00004.safetensors");
    load_safetensor_weights(llama3_model, "model_weights/model-00002-of-00004.safetensors");
    load_safetensor_weights(llama3_model, "model_weights/model-00003-of-00004.safetensors");
    load_safetensor_weights(llama3_model, "model_weights/model-00004-of-00004.safetensors");

    printf(WARN "[CPU]" RESET " Loaded model to CPU\n");
    printf(WARN "[CPU]" RESET " Moving model to CUDA\n");
    printf(WARN "[CPU]" RESET " Converting from BF16 to FP16\n");

    to_cuda(llama3_model);
    printf(GREEN "[CUDA]" RESET " Loaded to CUDA Device\n");
    printf(GREEN "[CUDA]" RESET " Formatted Parameters to FP16\n");

    // Load the tokenizer (this function should load the trie from the tokenizer's JSON)
    Llama3Tokenizer *llama3_tokenizer = load_tokenizer();
    if (llama3_tokenizer == NULL) {
        printf("Error: Failed to load the tokenizer\n");
        return 1;
    }

    CudaCache *Cache = init_cache(llama3_model);
    char *input_str = strdup("The Pacific Ocean is the largest ocean in the\n");

    /*
    char *input_str = (char *)malloc(sizeof(char) * 2048);
    fgets(input_str, 2048, stdin);
    */

    int *tokens = tokenize(llama3_tokenizer, input_str);
    if (tokens == NULL) {
        printf("Error: Tokenization failed\n");
        return 1;
    }

    Tensor *X = (Tensor *)malloc(sizeof(Tensor));
    int *d_tokens = tokens_to_cuda(tokens, 4096, X);
    printf(GREEN "[CUDA]" RESET " Tokenized input and moved to CUDA Device\n");

    inference(llama3_model, X, d_tokens, tokens, Cache);

    // Free the model resources
    free_llama3(llama3_model);

    return 0;
}