#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "inference/inference.cuh"
#include "llama3/llama3.cuh"
#include "safetensor/safetensor.cuh"
#include "tokenizer/tokenizer.cuh"

#define WARN "\033[1;33m"
#define GREEN "\033[1;32m"
#define GREY "\033[2m"
#define RESET "\033[0m"

#define CLEAR_TERMINAL() system("clear")

const int MODEL_NUM_LAYERS = 32;

extern int h_NUM_TOKENS;

int main() {
    // Initialize the Llama3 model
    Llama3 *llama3_model = init_llama3(MODEL_NUM_LAYERS);

    if (llama3_model == NULL) {
        printf("An error occurred while allocating memory for the Llama3 struct\n");
        exit(1);
    } else {
        printf("Model has been allocated with %d layers\n", llama3_model->n_layers);
    }

    // Load the safetensor weights into the model
    load_safetensor_weights(llama3_model, "model_weights/model-00001-of-00004.safetensors");
    load_safetensor_weights(llama3_model, "model_weights/model-00002-of-00004.safetensors");
    load_safetensor_weights(llama3_model, "model_weights/model-00003-of-00004.safetensors");
    load_safetensor_weights(llama3_model, "model_weights/model-00004-of-00004.safetensors");

    printf(WARN "[CPU]" RESET " Loaded model to CPU\n");
    printf(WARN "[CPU]" RESET " Moving model to CUDA\n");
    printf(WARN "[CPU]" RESET " Converting from BF16 to FP16\n");

    to_cuda(llama3_model);
    printf(GREEN "[CUDA]" RESET " Loaded to CUDA Device\n");
    printf(GREEN "[CUDA]" RESET " Formatted Parameters to FP16\n");

    // Load the tokenizer (this function should load the trie from the tokenizer's JSON)
    Llama3Tokenizer *llama3_tokenizer = load_tokenizer();
    if (llama3_tokenizer == NULL) {
        printf("Error: Failed to load the tokenizer\n");
        return 1;
    }

    CudaCache *Cache = init_cache(llama3_model);
    CLEAR_TERMINAL();
    while (true) {
        // char *input_str = strdup("<|begin_of_text|><|start_header_id|>system<|end_header_id|>\n\nYou are a helpful assistant, here to provide clear and concise answers to the user's questions.<|eot_id|><|start_header_id|>user<|end_header_id|>\n\nWhat is the largest ocean in the world?<|eot_id|><|start_header_id|>assistant<|end_header_id|>\n\n");
        char *input_str = strdup("What is the largest ocean in the world?\n\n");

        /*
        char *input_str = (char *)malloc(sizeof(char) * 2048);
        fgets(input_str, 2048, stdin);
        */

        int *tokens = tokenize(llama3_tokenizer, input_str);
        if (tokens == NULL) {
            printf("Error: Tokenization failed\n");
            return 1;
        }

        Tensor *X = (Tensor *)malloc(sizeof(Tensor));
        int *d_tokens = tokens_to_cuda(tokens, 4096, X);
        int next_token = 0;
        while (next_token < 128000) {
            next_token = inference(llama3_model, X, d_tokens, tokens, Cache);
            /*
            if (next_token < 128000) printf("%s\n", llama3_tokenizer->decode[next_token]);

            for (int i = 1; i < tokens[0]; i++) {
                printf("%d, ", tokens[i]);
            }
            printf("%d\n", tokens[0]);
            */
            if (tokens[0] > 2048) break;

            tokens[tokens[0]] = next_token;
            tokens[0] = tokens[0] + 1;

            /*
            for (int i = 1; i < tokens[0]; i++) {
                printf("%d, ", tokens[i]);
            }
            printf("%d\n", tokens[0]);
            */
            break;
        }
        free(tokens);
        _free_tensor(X);
        hipFree(d_tokens);

        exit(1);
    }

    // Free the model resources
    free_llama3(llama3_model);

    return 0;
}