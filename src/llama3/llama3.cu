#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "llama3.cuh"

Llama3 *init_llama3(int n_layers) {
    // Allocate memory for the Llama3 model
    Llama3 *llama3 = (Llama3 *)malloc(sizeof(Llama3));
    if (llama3 == NULL) {
        printf("Error: Memory allocation failed for Llama3 structure\n");
        exit(1);
    }

    llama3->n_layers = n_layers;

    // Allocate embed, norm, and lm_head tensors
    llama3->embed_tokens = (Tensor *)malloc(sizeof(Tensor));
    llama3->norm = (Tensor *)malloc(sizeof(Tensor));
    llama3->lm_head = (Tensor *)malloc(sizeof(Tensor));
    if (llama3->embed_tokens == NULL || llama3->norm == NULL || llama3->lm_head == NULL) {
        printf("Error: Memory allocation failed for embed, norm, or lm_head tensors\n");
        exit(1);
    }

    // Allocate and initialize each layer
    llama3->layers = (Llama3Layer **)malloc(sizeof(Llama3Layer *) * n_layers);
    if (llama3->layers == NULL) {
        printf("Error: Memory allocation failed for layers array\n");
        exit(1);
    }

    for (int i = 0; i < n_layers; i++) {
        Llama3Layer *layer = (Llama3Layer *)malloc(sizeof(Llama3Layer));
        if (layer == NULL) {
            printf("Error: Memory allocation failed for layer %d\n", i);
            exit(1);
        }

        // Initialize each tensor in the layer
        layer->input_layernorm = (Tensor *)malloc(sizeof(Tensor));
        layer->mlp_down_proj = (Tensor *)malloc(sizeof(Tensor));
        layer->mlp_gate_proj = (Tensor *)malloc(sizeof(Tensor));
        layer->mlp_up_proj = (Tensor *)malloc(sizeof(Tensor));
        layer->post_attention_layernorm = (Tensor *)malloc(sizeof(Tensor));
        layer->self_attn_k_proj = (Tensor *)malloc(sizeof(Tensor));
        layer->self_attn_o_proj = (Tensor *)malloc(sizeof(Tensor));
        layer->self_attn_q_proj = (Tensor *)malloc(sizeof(Tensor));
        layer->self_attn_v_proj = (Tensor *)malloc(sizeof(Tensor));

        layer->layernum = i;

        // Check for allocation failures for each tensor
        if (layer->input_layernorm == NULL || layer->mlp_down_proj == NULL ||
            layer->mlp_gate_proj == NULL || layer->mlp_up_proj == NULL ||
            layer->post_attention_layernorm == NULL || layer->self_attn_k_proj == NULL ||
            layer->self_attn_o_proj == NULL || layer->self_attn_q_proj == NULL ||
            layer->self_attn_v_proj == NULL) {
            printf("Error: Memory allocation failed for tensors in layer %d\n", i);
            exit(1);
        }

        llama3->layers[i] = layer;
    }

    return llama3;
}

void free_llama3(Llama3 *llama3) {
    if (!llama3) return;  // Ensure llama3 is not NULL before proceeding

    // Free non-layer tensors
    if (llama3->embed_tokens) {
        _free_tensor(llama3->embed_tokens);
        llama3->embed_tokens = NULL;
    }

    if (llama3->norm) {
        _free_tensor(llama3->norm);
        llama3->norm = NULL;
    }

    if (llama3->lm_head) {
        _free_tensor(llama3->lm_head);
        llama3->lm_head = NULL;
    }

    // Free each tensor inside the layers
    _m_component_tensor_operation(llama3, _free_tensor);

    // Free each layer structure
    for (int i = 0; i < llama3->n_layers; i++) {
        if (llama3->layers[i]) {
            free(llama3->layers[i]);
            llama3->layers[i] = NULL;  // Nullify after freeing
        }
    }

    // Free the layers array and the Llama3 structure
    if (llama3->layers) {
        free(llama3->layers);
        llama3->layers = NULL;  // Nullify after freeing
    }

    if (llama3) {
        free(llama3);
        llama3 = NULL;  // Nullify after freeing
    }
}

void _free_tensor(Tensor *tensor) {
    if (!tensor) return;  // Check if tensor is NULL before proceeding

    // Free CUDA memory, ensuring pointers are valid
    if (tensor->d_ndim) {
        hipFree(tensor->d_ndim);
        tensor->d_ndim = NULL;
    }
    if (tensor->d_mem_len) {
        hipFree(tensor->d_mem_len);
        tensor->d_mem_len = NULL;
    }
    if (tensor->d_shape) {
        hipFree(tensor->d_shape);
        tensor->d_shape = NULL;
    }
    if (tensor->d_fp16_tensor) {
        hipFree(tensor->d_fp16_tensor);
        tensor->d_fp16_tensor = NULL;
    }

    // Free CPU memory
    if (tensor->ndim) {
        free(tensor->ndim);
        tensor->ndim = NULL;
    }
    if (tensor->mem_len) {
        free(tensor->mem_len);
        tensor->mem_len = NULL;
    }
    if (tensor->shape) {
        free(tensor->shape);
        tensor->shape = NULL;
    }

    // Finally, free the Tensor structure itself
    free(tensor);
}

void to_cuda(Llama3 *llama3) {
    _m_component_tensor_operation(llama3, _preallocate_model_mem);
    _m_component_tensor_operation(llama3, _move_tensor_to_cuda);
}

void _cudaMalloc_fp16(Tensor *tensor) {
    __half *d_fp16_tensor;

    // Allocate fp16 tensor memory on the GPU
    hipMalloc((void **)&d_fp16_tensor, sizeof(__half) * (*(tensor->mem_len)));
    tensor->d_fp16_tensor = d_fp16_tensor;
}

void _preallocate_model_mem(Tensor *tensor) {
    int *d_ndim;
    int *d_mem_len;
    int *d_shape;
    __half *d_fp16_tensor;

    // Allocate GPU memory
    hipMalloc((void **)&d_ndim, sizeof(int));
    hipMalloc((void **)&d_mem_len, sizeof(int));
    hipMalloc((void **)&d_shape, sizeof(int) * (*(tensor->ndim)));
    hipMalloc((void **)&d_fp16_tensor, sizeof(__half) * (*(tensor->mem_len)));

    // Copy data from CPU to GPU
    hipMemcpy(d_ndim, tensor->ndim, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mem_len, tensor->mem_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_shape, tensor->shape, sizeof(int) * (*(tensor->ndim)), hipMemcpyHostToDevice);

    // Update tensor pointers to CUDA memory
    tensor->d_ndim = d_ndim;
    tensor->d_mem_len = d_mem_len;
    tensor->d_shape = d_shape;
    tensor->d_fp16_tensor = d_fp16_tensor;
}

void _move_tensor_to_cuda(Tensor *tensor) {
    uint16_t *d_bf16_tensor;

    // Allocate GPU memory
    hipMalloc((void **)&d_bf16_tensor, sizeof(uint16_t) * (*(tensor->mem_len)));

    // Copy data from CPU to GPU
    hipMemcpy(d_bf16_tensor, tensor->bf16_tensor, sizeof(uint16_t) * (*(tensor->mem_len)), hipMemcpyHostToDevice);

    // Free the CPU memory after transfer
    free(tensor->bf16_tensor);

    // Update tensor pointers to CUDA memory
    _kernel_wrapper_bf16_to_fp16(tensor);
}

void _kernel_wrapper_bf16_to_fp16(Tensor *tensor) {
    if (tensor->d_bf16_tensor == NULL) {
        printf("Error: Expected BF16 Tensor on Device to be allocated\n");
        exit(1);
    }

    // assign number of threads per block and blocks per grid
    int threads_per_block = 1024;
    int num_blocks = ((*(tensor->mem_len)) + threads_per_block - 1) / threads_per_block;

    _kernel_bf16_to_fp16<<<num_blocks, threads_per_block>>>(
        tensor->d_bf16_tensor, tensor->d_fp16_tensor, *(tensor->d_mem_len));
    hipDeviceSynchronize();

    // free unnnecessay tensor array after usage
    hipFree(tensor->d_bf16_tensor);
}

__global__ void _kernel_bf16_to_fp16(uint16_t *bf16_tensor, __half *fp16_tensor, int mem_len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < mem_len) {
        // Convert BF16 to FP32
        uint32_t bf16 = (uint32_t)bf16_tensor[idx];
        uint32_t fp32_bits = bf16 << 16;
        float fp32_value = __int_as_float(fp32_bits);

        // Convert FP32 to FP16
        fp16_tensor[idx] = __float2half_rn(fp32_value);
    }
}

// Applies a user-defined function to each tensor in the Llama3 model.
void _m_component_tensor_operation(Llama3 *llama3, void (*_func)(Tensor *)) {
    // perform singular function on all Tensors

    _func(llama3->embed_tokens);
    _func(llama3->lm_head);
    _func(llama3->norm);

    for (int i = 0; i < llama3->n_layers; i++) {
        _func(llama3->layers[i]->input_layernorm);
        _func(llama3->layers[i]->mlp_down_proj);
        _func(llama3->layers[i]->mlp_gate_proj);
        _func(llama3->layers[i]->mlp_up_proj);
        _func(llama3->layers[i]->post_attention_layernorm);
        _func(llama3->layers[i]->self_attn_k_proj);
        _func(llama3->layers[i]->self_attn_o_proj);
        _func(llama3->layers[i]->self_attn_q_proj);
        _func(llama3->layers[i]->self_attn_v_proj);
    }
}

int arr_to_mem_index(Tensor *t, int n, int *idx) {
    int mem_idx = 0;
    int stride = 1;

    // Compute the memory index using the tensor shape
    for (int i = n - 1; i >= 0; i--) {
        mem_idx += (idx[i] * stride);
        stride *= t->shape[i];
    }

    return mem_idx;
}
